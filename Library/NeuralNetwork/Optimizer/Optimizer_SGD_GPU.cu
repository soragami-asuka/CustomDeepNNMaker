//===============================================
// �œK�����[�`��(SGD)
//===============================================
#include"stdafx.h"

#include"Optimizer_SGD_base.h"

#pragma warning(push)
#pragma warning(disable : 4267)
#include <hip/hip_runtime.h> // need CUDA_VERSION
#include <hipDNN.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include ""
#pragma warning(pop)


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

	class Optimizer_SGD_GPU : public Optimizer_SGD_base
	{
	private:
		hipblasHandle_t cublasHandle;

	public:
		/** �R���X�g���N�^ */
		Optimizer_SGD_GPU(U32 i_parameterCount)
			:	Optimizer_SGD_base	(i_parameterCount)
		{
			hipblasCreate(&cublasHandle);
		}
		/** �f�X�g���N�^ */
		virtual ~Optimizer_SGD_GPU()
		{
			hipblasDestroy(cublasHandle);
		}

	public:
		//===========================
		// ����
		//===========================
		/** �p�����[�^���X�V����.
			@param io_lpParamter	�X�V����p�����[�^.
			@param io_lpDParameter	�p�����[�^�̕ω���. */
		ErrorCode UpdateParameter(F32 io_lpParameter[], const F32 i_lpDParameter[])
		{
			hipblasSaxpy(
				this->cublasHandle,
				this->m_parameterCount,
				&this->m_learnCoeff,
				i_lpDParameter,
				1,
				io_lpParameter,
				1);

			return ErrorCode::ERROR_CODE_NONE;
		}
	};

	/** �I�v�e�B�}�C�U���쐬���� */
	IOptimizer* CreateOptimizer_SGD_GPU(U32 i_parameterCount)
	{
		return new Optimizer_SGD_GPU(i_parameterCount);
	}
	/** �I�v�e�B�}�C�U���o�b�t�@����쐬���� */
	IOptimizer* CreateOptimizerFromBuffer_SGD_GPU(const BYTE* i_lpBuffer, Gravisbell::S32 i_bufferSize, Gravisbell::S32& o_useBufferSize)
	{
		return CreateOptimizerFromBuffer_SGD(i_lpBuffer, i_bufferSize, o_useBufferSize, CreateOptimizer_SGD_GPU);
	}
	/** �I�v�e�B�}�C�U�[���X�V����.�قȂ�^�������ꍇ�͋����I�Ɏw��̌^�ɕϊ������. */
	ErrorCode ChangeOptimizer_SGD_GPU(IOptimizer** io_ppOptimizer, U32 i_parameterCount)
	{
		Optimizer_SGD_GPU* pOptimizer = dynamic_cast<Optimizer_SGD_GPU*>(*io_ppOptimizer);
		if(pOptimizer == NULL)
		{
			if(*io_ppOptimizer)
				delete *io_ppOptimizer;

			*io_ppOptimizer = CreateOptimizer_SGD_GPU(i_parameterCount);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

}	// NeuralNetwork
}	// Layer
}	// Gravisbell