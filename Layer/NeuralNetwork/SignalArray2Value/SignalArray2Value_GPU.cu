#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"SignalArray2Value_DATA.hpp"
#include"SignalArray2Value_FUNC.hpp"
#include"SignalArray2Value_Base.h"

#include"SignalArray2Value_GPU.cuh"
#include"SignalArray2Value_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {
	
#define THREAD_PER_BLOCK	32

	/** ���͂𑫂����킹��.
		<outputChNo, batchSize> <32>
		*/
	__global__ void device_SignalArray2Value(
		F32* o_lpOutput,
		const F32* i_lpInput,
		U32 i_resolution, U32 i_bufferPerCh, U32 i_loopCount,
		F32 outputMinValue,
		F32 outputMaxValue)
	{
		U32 batchNum = blockIdx.y;
		U32 outputChNo = blockIdx.x;
		U32 outputChCount = gridDim.x;
		U32 tid = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = loopNum * THREAD_PER_BLOCK + tid;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 outputOffset = (batchNum * outputChCount + outputChNo) * i_bufferPerCh + bufferPos;

			// �ő�l�����߂�
			U32 maxNum = 0;
			F32 maxValue = -FLT_MAX;
			for(U32 inputNum=0; inputNum<i_resolution; inputNum++)
			{
				U32 inputChNum = outputChNo * i_resolution + inputNum;
				U32 inputOffset = (batchNum * (outputChCount * i_resolution) + outputChNo * i_resolution + inputChNum) * i_bufferPerCh + bufferPos;

				F32 value = i_lpInput[inputOffset];

				if(value > maxValue)
				{
					maxNum = inputNum;
					maxValue = value;
				}
			}

			o_lpOutput[outputOffset] = ((F32)maxNum / (i_resolution-1)) * (outputMaxValue - outputMinValue) + outputMinValue;
		}
	}

	__global__ void device_Value2SignalArray(
		U32 inputChSize,
		U32 outputBatchBufferSize,
		F32 outputMinValue,
		F32 outputMaxValue,
		F32 lpDInputBuffer[],
		const F32 lpOutputBuffer[],
		const F32 lpDOutputBuffer[],
		U32 i_loopCount,
		U32 i_bufferPerCh)
	{
		U32 batchNum  = blockIdx.x;
		U32 tid          = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = loopNum * THREAD_PER_BLOCK + tid;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 outputOffset = outputBatchBufferSize * batchNum + bufferPos;

			F32 teachValue = lpOutputBuffer[outputOffset] + lpDOutputBuffer[outputOffset];

			U32 teachCh = max(0, min(inputChSize-1, (U32)((teachValue - outputMinValue) / (outputMaxValue - outputMinValue) * (inputChSize-1) + 0.5f)));

			U32 inputOffset = (i_bufferPerCh * inputChSize * batchNum) + (i_bufferPerCh * teachCh) + bufferPos;

			lpDInputBuffer[inputOffset] = 1.0f;
		}
	}


	/** �R���X�g���N�^ */
	SignalArray2Value_GPU::SignalArray2Value_GPU(Gravisbell::GUID guid, SignalArray2Value_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	SignalArray2Value_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	SignalArray2Value_GPU::~SignalArray2Value_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 SignalArray2Value_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode SignalArray2Value_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	SignalArray2Value_LayerData_Base& SignalArray2Value_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const SignalArray2Value_LayerData_Base& SignalArray2Value_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode SignalArray2Value_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode SignalArray2Value_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// ���͐M���̃`�����l�����Ƃ̃o�b�t�@�T�C�Y
		this->bufferPerChannel = this->GetOutputDataStruct().x * this->GetOutputDataStruct().y * this->GetOutputDataStruct().z;

		/**< ���͐M���̃o�b�`���Ƃ̃o�b�t�@�T�C�Y */
		this->inputBatchBufferSize = this->bufferPerChannel * this->GetInputDataStruct().ch;

		// �ꎞ�o�̓o�b�t�@(�z�X�g������)
		this->lpTmpOutputBuffer_h.resize(this->outputBufferCount * this->GetBatchSize());
		this->lpTmpBatchOutputBuffer_h.resize(this->GetBatchSize());
		for(U32 i=0; i<this->GetBatchSize(); i++)
			this->lpTmpBatchOutputBuffer_h[i] = &this->lpTmpOutputBuffer_h[this->outputBufferCount * i];

		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode SignalArray2Value_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode SignalArray2Value_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// �o�̓o�b�t�@�̏�����
		hipMemset(o_lppOutputBuffer, 0, sizeof(F32)*this->outputBufferCount*this->GetBatchSize());
		memset(&this->lpTmpOutputBuffer_h[0], 0, sizeof(F32)*this->lpTmpOutputBuffer_h.size());

#if 0
		// �v�Z
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
		{
			for(U32 z=0; z<this->GetInputDataStruct().z; z++)
			{
				for(U32 y=0; y<this->GetInputDataStruct().y; y++)
				{
					for(U32 x=0; x<this->GetInputDataStruct().x; x++)
					{
						U32 offset = this->GetInputDataStruct().POSITION_TO_OFFSET(x, y, z, 0);

						// �ő�l�̔ԍ����擾
						S32 maxPos = -1;
						hipblasIsamax(
							this->cublasHandle,
							this->inputBatchBufferSize,
							&i_lppInputBuffer[this->inputBatchBufferSize*batchNum + offset],
							this->bufferPerChannel,
							&maxPos);

						if(maxPos <= 0)
							continue;
						// maxPos��1�`�Ȃ̂ŁA0�`�ɏ���������
						maxPos-=1;

						this->lpTmpBatchOutputBuffer_h[batchNum][offset]
							= (F32)maxPos / (this->GetInputDataStruct().ch -1)
							* (this->layerData.layerStructure.outputMaxValue - this->layerData.layerStructure.outputMinValue)
							+ this->layerData.layerStructure.outputMinValue;
					}
				}
			}
		}

		// CPU > GPU
		hipMemcpy(
			o_lppOutputBuffer,
			&this->lpTmpOutputBuffer_h[0],
			sizeof(F32) * this->lpTmpOutputBuffer_h.size(),
			hipMemcpyHostToDevice);
#else
		dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->bufferPerChannel + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		device_SignalArray2Value<<<grid, block>>>(
			o_lppOutputBuffer,
			i_lppInputBuffer,
			this->layerData.layerStructure.resolution,
			this->bufferPerChannel,
			loopCount,
			this->layerData.layerStructure.outputMinValue, this->layerData.layerStructure.outputMaxValue);

#endif

#if _DEBUG
			std::vector<F32> lpInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpInputBuffer[0], i_lppInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], o_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode SignalArray2Value_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// �o�̓o�b�t�@�̏�����
			hipMemset(o_lppDInputBuffer, 0, sizeof(F32)*this->inputBufferCount*this->GetBatchSize());

			dim3 grid(this->GetBatchSize());
			dim3 block(THREAD_PER_BLOCK);
			U32 loopCount = (this->bufferPerChannel + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

			// �o�͌덷�o�b�t�@�ɐ���������������
			device_Value2SignalArray<<<grid, block>>>(
				this->GetInputDataStruct().ch,
				this->outputBufferCount,
				this->layerData.layerStructure.outputMinValue,
				this->layerData.layerStructure.outputMaxValue,
				o_lppDInputBuffer,
				i_lppOutputBuffer,
				i_lppDOutputBuffer,
				loopCount,
				this->bufferPerChannel);

#if _DEBUG
			std::vector<F32> lpDOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpTeachBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpTeachBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			// �����Əo�͂Ō덷�����
			F32 alpha = -1;
			hipblasSaxpy(
				this->cublasHandle,
				this->inputBufferCount * this->GetBatchSize(),
				&alpha,
				i_lppInputBuffer,
				1,
				o_lppDInputBuffer,
				1);

#if _DEBUG
			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], i_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpDInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDInputBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode SignalArray2Value_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
