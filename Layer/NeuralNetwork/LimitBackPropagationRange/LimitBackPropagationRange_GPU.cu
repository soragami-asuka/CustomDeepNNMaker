#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"LimitBackPropagationRange_DATA.hpp"
#include"LimitBackPropagationRange_FUNC.hpp"
#include"LimitBackPropagationRange_Base.h"

#include"LimitBackPropagationRange_GPU.cuh"
#include"LimitBackPropagationRange_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {
	
#define THREAD_PER_BLOCK	32
#define CALC_BATCH_MAX	(256)
#define CALC_INPUT_MAX	(1024)
	
	__global__ void device_ReLimitBackPropagationRange(
		U32 chCount,
		U32 startX, U32 startY, U32 startZ,
		U32 inputXCount,  U32 inputYCount,  U32 inputZCount,
		U32 outputXCount, U32 outputYCount, U32 outputZCount,
		U32 i_bufferPerCh,
		U32 i_loopCount,
		const F32 lpDOutputBuffer[],
		F32 lpDInputBuffer[])
	{
		U32 chNum    = blockIdx.x;
		U32 batchNum = blockIdx.y;
		U32 tid = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = tid*i_loopCount + loopNum;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 x = bufferPos % inputXCount;
			U32 y = bufferPos / inputXCount % inputYCount;
			U32 z = bufferPos / inputXCount / inputYCount % inputZCount;

			if(x < startX || x >= startX + outputXCount)
				continue;
			if(y < startY || y >= startY + outputYCount)
				continue;
			if(z < startZ || z >= startZ + outputZCount)
				continue;

			U32 offset = batchNum * (chCount * i_bufferPerCh) + chNum * i_bufferPerCh + bufferPos;

			lpDInputBuffer[offset] = lpDOutputBuffer[offset];
		}
	}

	/** �R���X�g���N�^ */
	LimitBackPropagationRange_GPU::LimitBackPropagationRange_GPU(Gravisbell::GUID guid, LimitBackPropagationRange_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	LimitBackPropagationRange_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	LimitBackPropagationRange_GPU::~LimitBackPropagationRange_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 LimitBackPropagationRange_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode LimitBackPropagationRange_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	LimitBackPropagationRange_LayerData_Base& LimitBackPropagationRange_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const LimitBackPropagationRange_LayerData_Base& LimitBackPropagationRange_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode LimitBackPropagationRange_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode LimitBackPropagationRange_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// CH������̃o�b�t�@��
		this->bufferCountPerCh = this->GetOutputDataStruct().x * this->GetOutputDataStruct().y * this->GetOutputDataStruct().z;

		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode LimitBackPropagationRange_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode LimitBackPropagationRange_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// ���̓o�b�t�@���o�̓o�b�t�@�ɃR�s�[
		hipMemcpy(o_lppOutputBuffer, i_lppInputBuffer, sizeof(F32)*this->GetBatchSize()*this->outputBufferCount, hipMemcpyDeviceToDevice);

#if _DEBUG
			std::vector<F32> lpInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpInputBuffer[0], i_lppInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], o_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode LimitBackPropagationRange_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// �o�̓o�b�t�@�̏�����
			hipMemset(o_lppDInputBuffer, 0, sizeof(F32)*this->inputBufferCount*this->GetBatchSize());
			
			// �v�Z
			dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
			dim3 block(THREAD_PER_BLOCK);
			U32 loopCount = (this->bufferCountPerCh + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

			device_ReLimitBackPropagationRange<<<grid, block>>>(
				this->GetOutputDataStruct().ch,
				this->layerData.layerStructure.startPosition.x, this->layerData.layerStructure.startPosition.y, this->layerData.layerStructure.startPosition.z,
				this->GetInputDataStruct().x, this->GetInputDataStruct().y, this->GetInputDataStruct().z,
				this->layerData.layerStructure.boxSize.x, this->layerData.layerStructure.boxSize.y, this->layerData.layerStructure.boxSize.z,
				this->bufferCountPerCh, loopCount,
				i_lppDOutputBuffer,
				o_lppDInputBuffer);
			
#if _DEBUG
			std::vector<F32> lpDInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDInputBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpDOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode LimitBackPropagationRange_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
