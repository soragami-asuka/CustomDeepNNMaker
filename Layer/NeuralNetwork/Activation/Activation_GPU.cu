#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// CPU�����p
//======================================
#include"stdafx.h"

#include"Activation_DATA.hpp"
#include"Activation_FUNC.hpp"
#include"Activation_Base.h"

#include"Activation_GPU.cuh"
#include"Activation_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define BLOCK_SIZE	(32)

namespace
{
	//===========================
	// Leaky-ReLU
	//===========================
	__global__ void cuda_func_activation_LeakyReLU(const F32* i_lpInputBuffer, F32* o_lpOutputBuffer, F32 i_alpha, U32 i_bufferSize)
	{
		const U32 inputNum = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		if(inputNum >= i_bufferSize)	// ���򂷂邪������warp�����Ȃ̂ŁA�������x�ɉe���͂Ȃ��͂�...
			return;

		o_lpOutputBuffer[inputNum] = i_lpInputBuffer[inputNum] * ((i_lpInputBuffer[inputNum]>0) + i_alpha * (i_lpInputBuffer[inputNum]<=0));
	}
	__global__ void cuda_func_dactivation_LeakyReLU(const F32* i_lpOutputBuffer, const F32* i_lpDOutputBuffer, F32* o_lpOutputBuffer, F32 i_alpha, U32 i_bufferSize)
	{
		const U32 inputNum = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		if(inputNum >= i_bufferSize)	// ���򂷂邪������warp�����Ȃ̂ŁA�������x�ɉe���͂Ȃ��͂�...
			return;

		o_lpOutputBuffer[inputNum] = ((i_lpOutputBuffer[inputNum]>0) + i_alpha * (i_lpOutputBuffer[inputNum]<=0)) * i_lpDOutputBuffer[inputNum];
	}
}


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	Activation_GPU::Activation_GPU(Gravisbell::GUID guid, Activation_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct)
		:	Activation_Base	(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)		/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)		/**< �o�̓o�b�t�@�� */
		,	cudnnHandle		(NULL)
		,	activDesc		(NULL)
		,	inputTensorDesc	(NULL)
		,	outputTensorDesc	(NULL)
	{
		hipdnnCreate(&cudnnHandle);
		hipdnnCreateTensorDescriptor(&inputTensorDesc);
		hipdnnCreateTensorDescriptor(&outputTensorDesc);
		hipdnnCreateActivationDescriptor(&activDesc);
	}
	/** �f�X�g���N�^ */
	Activation_GPU::~Activation_GPU()
	{
		if(inputTensorDesc)		hipdnnDestroyTensorDescriptor(inputTensorDesc);
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(outputTensorDesc);
		if(activDesc)			hipdnnDestroyActivationDescriptor(activDesc);
		if(cudnnHandle)			hipdnnDestroy(cudnnHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 Activation_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode Activation_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	ILayerData& Activation_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const ILayerData& Activation_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �o�̓o�b�t�@���쐬
		switch(this->layerData.layerStructure.ActivationType)
		{
			// lenear
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_lenear:
			break;

		default:
			this->lpOutputBuffer_d.resize(this->GetBatchSize() * this->outputBufferCount);
			{
				int n = this->GetBatchSize();
				int c = this->GetOutputDataStruct().ch;
				int h = this->GetOutputDataStruct().z * this->GetOutputDataStruct().y;
				int w = this->GetOutputDataStruct().x;

				const int nDims = 4;
				int dimA[nDims] = {n, c, h, w};
				int strideA[nDims] = {c*h*w, h*w, w, 1};

				hipdnnStatus_t err = hipdnnSetTensorNdDescriptor(this->outputTensorDesc,
					HIPDNN_DATA_FLOAT,
					4,
					dimA,
					strideA );

				if(err != 0)
					break;

				err = hipdnnSetTensorNdDescriptor(this->inputTensorDesc,
					HIPDNN_DATA_FLOAT,
					4,
					dimA,
					strideA );

				if(err != 0)
					break;
			}
			break;
		}


		// �������֐���ݒ�
		switch(this->layerData.layerStructure.ActivationType)
		{
			// lenear
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_lenear:
			break;

			// Sigmoid
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid_crossEntropy:
		default:
			hipdnnSetActivationDescriptor(activDesc,
										HIPDNN_ACTIVATION_SIGMOID,
										HIPDNN_PROPAGATE_NAN,
										0.0);
			break;

			// ReLU
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_ReLU:
			hipdnnSetActivationDescriptor(activDesc,
										HIPDNN_ACTIVATION_RELU,
										HIPDNN_PROPAGATE_NAN,
										0.0);
			break;

			// Leaky-ReLU
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_LeakyReLU:
			break;

			// tanh
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_tanh:
			hipdnnSetActivationDescriptor(activDesc,
										HIPDNN_ACTIVATION_TANH,
										HIPDNN_PROPAGATE_NAN,
										0.0);
			break;

			// SoftMax�n
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH:
			break;
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL_crossEntropy:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH_crossEntropy:
			break;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	
	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_GPU::PreProcessLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode Activation_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		// ���̓o�b�t�@�̃A�h���X��z��Ɋi�[
		this->m_lpInputBuffer_d = i_lpInputBuffer;


		switch(this->layerData.layerStructure.ActivationType)
		{
			// lenear
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_lenear:
			break;

		default:
			// Sigmoid
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid_crossEntropy:
			// ReLU
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_ReLU:
			// tanh
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_tanh:
			{
				F32 alpha = 1.0f;
				F32 beta = 0.0f;
				hipdnnActivationForward(
					this->cudnnHandle,
					this->activDesc,
					&alpha,
					inputTensorDesc,
					this->m_lpInputBuffer_d,
					&beta,
					outputTensorDesc,
					thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]));
			}
			break;

			// Leaky-ReLU
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_LeakyReLU:
			{
				U32 MAX_BUFFER_SIZE = 32768;
				U32 bufferSize = this->inputBufferCount * this->GetBatchSize();
				U32 remainingSize = bufferSize;
				while(remainingSize > 0)
				{
					U32 bufferCount = min(remainingSize, MAX_BUFFER_SIZE);
					dim3 grid((bufferCount +(BLOCK_SIZE - 1))/BLOCK_SIZE , 1, 1);
					dim3 block(BLOCK_SIZE, 1, 1);

					U32 offset = bufferSize - remainingSize;
					
					cuda_func_activation_LeakyReLU<<<grid, block>>>(
						&i_lpInputBuffer[offset],
						thrust::raw_pointer_cast(&this->lpOutputBuffer_d[offset]),
						this->layerData.layerStructure.LeakyReLU_alpha,
						bufferCount);

					remainingSize = max(0, (S32)remainingSize-(S32)MAX_BUFFER_SIZE);
				}
			}	
			break;

			// softmax
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL_crossEntropy:
			{
				F32 alpha = 1.0f;
				F32 beta = 0.0f;
				hipdnnStatus_t err =	hipdnnSoftmaxForward(
					this->cudnnHandle,
					HIPDNN_SOFTMAX_ACCURATE,
					HIPDNN_SOFTMAX_MODE_INSTANCE,
					&alpha,
					this->inputTensorDesc,
					this->m_lpInputBuffer_d,
					&beta,
					this->outputTensorDesc,
					thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]));
			}
			break;
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH:
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH_crossEntropy:
			{
				F32 alpha = 1.0f;
				F32 beta = 0.0f;
				hipdnnSoftmaxForward(
					this->cudnnHandle,
					HIPDNN_SOFTMAX_ACCURATE,
					HIPDNN_SOFTMAX_MODE_CHANNEL,
					&alpha,
					this->inputTensorDesc,
					this->m_lpInputBuffer_d,
					&beta,
					this->outputTensorDesc,
					thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]));
			}
			break;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER Activation_GPU::GetOutputBuffer()const
	{
		switch(this->layerData.layerStructure.ActivationType)
		{
			// lenear
		case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_lenear:
			return this->m_lpInputBuffer_d;

		default:
			return thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]);
		}
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode Activation_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32)*outputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Activation_GPU::CalculateDInput(BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// �o�͌덷�o�b�t�@�̃A�h���X��z��Ɋi�[
		this->m_lpDOutputBufferPrev_d = i_lppDOutputBuffer;


		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// ���͌덷�o�b�t�@�̃A�h���X���i�[
			this->m_lpDInputBuffer_d = o_lppDInputBuffer;

			switch(this->layerData.layerStructure.ActivationType)
			{
				// lenear
			case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_lenear:
				hipMemcpy(this->m_lpDInputBuffer_d, this->m_lpDOutputBufferPrev_d, sizeof(F32)*this->inputBufferCount*this->GetBatchSize(), hipMemcpyDeviceToDevice);
				break;

			default:
				// Sigmoid
			case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid:
				// ReLU
			case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_ReLU:
				// tanh
			case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_tanh:
				{
					F32 alpha = 1.0f;
					F32 beta = 0.0f;
					hipdnnActivationBackward(
						this->cudnnHandle,
						this->activDesc,
						&alpha,
						this->outputTensorDesc,	// �o�̓f�[�^�\��
						thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]),	// �o�̓f�[�^
						this->outputTensorDesc,
						this->m_lpDOutputBufferPrev_d,	// �o�͌덷
						this->inputTensorDesc,
						this->m_lpInputBuffer_d,	// ����
						&beta,
						this->inputTensorDesc,
						this->m_lpDInputBuffer_d	// ���͌덷
						);
				}
				break;

					// Leaky-ReLU
				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_LeakyReLU:
					{
						U32 MAX_BUFFER_SIZE = 32768;
						U32 bufferSize = this->inputBufferCount * this->GetBatchSize();
						U32 remainingSize = bufferSize;
						while(remainingSize > 0)
						{
							U32 bufferCount = min(remainingSize, MAX_BUFFER_SIZE);
							dim3 grid((bufferCount +(BLOCK_SIZE - 1))/BLOCK_SIZE , 1, 1);
							dim3 block(BLOCK_SIZE, 1, 1);

							U32 offset = bufferSize - remainingSize;

							cuda_func_dactivation_LeakyReLU<<<grid, block>>>(
								thrust::raw_pointer_cast(&this->lpOutputBuffer_d[offset]),
								&i_lppDOutputBuffer[offset],
								&o_lppDInputBuffer[offset],
								this->layerData.layerStructure.LeakyReLU_alpha,
								bufferCount);

							remainingSize = max(0, (S32)remainingSize-(S32)MAX_BUFFER_SIZE);
						}
					}
					break;

				// softmax
				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL:
					{
						F32 alpha = 1.0f;
						F32 beta = 0.0f;
						hipdnnSoftmaxBackward(
							this->cudnnHandle,
							HIPDNN_SOFTMAX_ACCURATE,
							HIPDNN_SOFTMAX_MODE_INSTANCE,
							&alpha,
							this->outputTensorDesc,
							thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]),
							this->outputTensorDesc,
							this->m_lpDOutputBufferPrev_d,
							&beta,
							this->inputTensorDesc,
							this->m_lpDInputBuffer_d
							);
					}
					break;
				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH:
					{
						F32 alpha = 1.0f;
						F32 beta = 0.0f;
						hipdnnSoftmaxBackward(
							this->cudnnHandle,
							HIPDNN_SOFTMAX_ACCURATE,
							HIPDNN_SOFTMAX_MODE_CHANNEL,
							&alpha,
							this->outputTensorDesc,
							thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]),
							this->outputTensorDesc,
							this->m_lpDOutputBufferPrev_d,
							&beta,
							this->inputTensorDesc,
							this->m_lpDInputBuffer_d
							);
					}
					break;

				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_sigmoid_crossEntropy:
				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_ALL_crossEntropy:
				case Gravisbell::Layer::NeuralNetwork::Activation::LayerStructure::ActivationType_softmax_CH_crossEntropy:
					hipMemcpy(this->m_lpDInputBuffer_d, this->m_lpDOutputBufferPrev_d, sizeof(F32)*this->inputBufferCount*this->GetBatchSize(), hipMemcpyDeviceToDevice);
					break;
			}
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K�덷���v�Z����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Activation_GPU::Training(BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput(o_lppDInputBuffer, i_lppDOutputBuffer);
	}


	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER Activation_GPU::GetDInputBuffer()const
	{
		return this->m_lpDInputBuffer_d;
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode Activation_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32)*inputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
