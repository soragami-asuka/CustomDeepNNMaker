//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"Reshape_DATA.hpp"
#include"Reshape_FUNC.hpp"
#include"Reshape_Base.h"

#include"Reshape_GPU.cuh"
#include"Reshape_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	Reshape_GPU::Reshape_GPU(Gravisbell::GUID guid, Reshape_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	Reshape_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
	{
	}
	/** �f�X�g���N�^ */
	Reshape_GPU::~Reshape_GPU()
	{
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 Reshape_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode Reshape_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	Reshape_LayerData_Base& Reshape_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const Reshape_LayerData_Base& Reshape_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_GPU::PreProcessCalculate()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode Reshape_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		hipMemcpy(o_lppOutputBuffer, i_lppInputBuffer, sizeof(F32)*this->GetInputBufferCount()*this->GetBatchSize(), hipMemcpyDeviceToDevice);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Reshape_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			hipMemcpy(
				o_lppDInputBuffer,
				i_lppDOutputBuffer,
				sizeof(F32)*this->GetInputBufferCount()*this->GetBatchSize(),
				hipMemcpyDeviceToDevice);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Reshape_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
