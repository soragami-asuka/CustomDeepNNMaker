#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"GlobalAveragePooling_DATA.hpp"
#include"GlobalAveragePooling_FUNC.hpp"
#include"GlobalAveragePooling_Base.h"

#include"GlobalAveragePooling_GPU.cuh"
#include"GlobalAveragePooling_LayerData_GPU.cuh"

#include<device_functions.hpp>

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define BLOCK_SIZE	(32)

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

	namespace
	{
		// ����p
		__global__ void cuda_func_average(const F32* i_lpInputBuffer, F32* o_lpOutputBuffer, const U32 i_inputChSize, U32 i_outputChSize)
		{
			const U32 batchNo = blockIdx.z;
			const U32 chNo    = blockIdx.y;
			const U32 chCount = gridDim.y;

			const U32 bufferPos = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			const U32 outputPos = batchNo * (i_outputChSize * chCount) + chNo * i_outputChSize + blockIdx.x;
			const U32 inputPos  = batchNo * (i_inputChSize  * chCount) + chNo * i_inputChSize  + bufferPos;

			__shared__ F32 lpTmpBuf[BLOCK_SIZE*2];
			if(bufferPos >= i_inputChSize)
				lpTmpBuf[threadIdx.x]  = 0.0f;
			else
				lpTmpBuf[threadIdx.x]  = i_lpInputBuffer[inputPos];
			__syncthreads();

			if(threadIdx.x < 16)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 16];
			__syncthreads();
			if(threadIdx.x < 8)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 8];
			__syncthreads();
			if(threadIdx.x < 4)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 4];
			__syncthreads();
			if(threadIdx.x < 2)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 2];
			__syncthreads();
			if(threadIdx.x < 1)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 1];
			__syncthreads();

			if(threadIdx.x < 1)
				o_lpOutputBuffer[outputPos] = lpTmpBuf[0];
		}


		// �o�͌덷����͌덷�ɕϊ�����
		__global__ void cuda_func_DOutput_to_DInput(const F32* i_lpDOutputBuffer, F32* o_lpDInputBuffer, const U32 i_inputChSize)
		{
			const U32 batchNo = blockIdx.z;
			const U32 chNo    = blockIdx.y;
			const U32 chCount = gridDim.y;

			const U32 inpuBufferPos   = blockIdx.x * BLOCK_SIZE + threadIdx.x;
			
			const U32 inputPos  = batchNo * (chCount * i_inputChSize) + chNo * i_inputChSize + inpuBufferPos;
			const U32 outputPos = batchNo *  chCount + chNo;


			if(inpuBufferPos < i_inputChSize)
			{
				o_lpDInputBuffer[inputPos] = i_lpDOutputBuffer[outputPos] / i_inputChSize;
			}
		}
	}


	/** �R���X�g���N�^ */
	GlobalAveragePooling_GPU::GlobalAveragePooling_GPU(Gravisbell::GUID guid, GlobalAveragePooling_LayerData_GPU& i_layerData)
		:	GlobalAveragePooling_Base	(guid)
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
		,	m_lppInputBuffer				(NULL)			/**< ���Z���̓��̓f�[�^ */
		,	m_lppDOutputBufferPrev			(NULL)			/**< ���͌덷�v�Z���̏o�͌덷�f�[�^ */
		,	cublasHandle					(NULL)
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	GlobalAveragePooling_GPU::~GlobalAveragePooling_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 GlobalAveragePooling_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode GlobalAveragePooling_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	GlobalAveragePooling_LayerData_Base& GlobalAveragePooling_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const GlobalAveragePooling_LayerData_Base& GlobalAveragePooling_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode GlobalAveragePooling_GPU::PreProcessLearn(unsigned int batchSize)
	{
		ErrorCode errorCode = this->PreProcessCalculate(batchSize);
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// ���͍����o�b�t�@���쐬
		this->lpDInputBuffer.resize(this->batchSize * this->inputBufferCount);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode GlobalAveragePooling_GPU::PreProcessCalculate(unsigned int batchSize)
	{
		this->batchSize = batchSize;

		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �o�̓o�b�t�@���쐬
		this->lpOutputBuffer.resize(this->batchSize * this->outputBufferCount);

		// 1CH������̃T�C�Y���v�Z
		this->chSize = this->GetInputDataStruct().x * this->GetInputDataStruct().y * this->GetInputDataStruct().z;

		// �ꎞ�o�b�t�@�̊m��
		this->lpTmpBuffer0.resize((this->chSize + 31)/32*32 * this->GetInputDataStruct().ch * this->batchSize, 0.0f);
		this->lpTmpBuffer1.resize((this->chSize + 31)/32*32 * this->GetInputDataStruct().ch * this->batchSize, 0.0f);
		this->lpTmpOutputBuffer_host.resize(this->outputBufferCount * this->batchSize);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K���[�v�̏���������.�f�[�^�Z�b�g�̊w�K�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode GlobalAveragePooling_GPU::PreProcessLearnLoop(const SettingData::Standard::IData& data)
	{
		if(this->pLearnData != NULL)
			delete this->pLearnData;
		this->pLearnData = data.Clone();

		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}
	/** ���Z���[�v�̏���������.�f�[�^�Z�b�g�̉��Z�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode GlobalAveragePooling_GPU::PreProcessCalculateLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode GlobalAveragePooling_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		// ���̓o�b�t�@�̃A�h���X���i�[
		this->m_lppInputBuffer = i_lpInputBuffer;

#ifdef _DEBUG
		std::vector<F32> lpInputBuffer_host(this->inputBufferCount * this->batchSize);
		hipMemcpy(&lpInputBuffer_host[0], i_lpInputBuffer, sizeof(F32)*lpInputBuffer_host.size(), hipMemcpyDeviceToHost);
#endif

		// ���񏈗�
		U32 tmpInputBufferCount = this->chSize;
		U32 tmpOutputBufferCount = (tmpInputBufferCount + (BLOCK_SIZE-1))/BLOCK_SIZE;
		{
			dim3 grid(tmpOutputBufferCount, this->GetInputDataStruct().ch, this->batchSize);

			cuda_func_average<<<grid, BLOCK_SIZE>>>(i_lpInputBuffer, thrust::raw_pointer_cast(&this->lpTmpBuffer0[0]), tmpInputBufferCount, tmpOutputBufferCount);
		}
		thrust::device_vector<F32>* pTmpBufferIn  = &this->lpTmpBuffer0;
		thrust::device_vector<F32>* pTmpBufferOut = &this->lpTmpBuffer1;


#ifdef _DEBUG
		std::vector<F32> lpTmpBuffer_host(tmpOutputBufferCount * this->GetInputDataStruct().ch * this->batchSize);
		hipMemcpy(&lpTmpBuffer_host[0], thrust::raw_pointer_cast(&(*pTmpBufferIn)[0]), sizeof(F32)*lpTmpBuffer_host.size(), hipMemcpyDeviceToHost);
#endif


		while(tmpOutputBufferCount > 1)
		{
			tmpInputBufferCount = tmpOutputBufferCount;
			tmpOutputBufferCount = (tmpInputBufferCount + (BLOCK_SIZE-1))/BLOCK_SIZE;

			dim3 grid(tmpOutputBufferCount, this->GetInputDataStruct().ch, this->batchSize);

			cuda_func_average<<<grid, BLOCK_SIZE>>>(
				thrust::raw_pointer_cast(&(*pTmpBufferIn)[0]),
				thrust::raw_pointer_cast(&(*pTmpBufferOut)[0]),
				tmpInputBufferCount, tmpOutputBufferCount);

			thrust::device_vector<F32>* pTmpBufferTmp = pTmpBufferIn;
			pTmpBufferIn  = pTmpBufferOut;
			pTmpBufferOut = pTmpBufferTmp;
		}

		// �eCH�̗v�f��ch�T�C�Y�ŏ��Z���Ė{�̂Ɋi�[
		hipMemcpy(
			thrust::raw_pointer_cast(&this->lpTmpOutputBuffer_host[0]),
			thrust::raw_pointer_cast(&(*pTmpBufferIn)[0]),
			sizeof(F32)*this->outputBufferCount*this->batchSize,
			hipMemcpyDeviceToHost);
		for(U32 outputNum=0; outputNum<this->lpOutputBuffer.size(); outputNum++)
		{
			lpTmpOutputBuffer_host[outputNum] /= this->chSize;
		}
		this->lpOutputBuffer = lpTmpOutputBuffer_host;
		
#ifdef _DEBUG
		std::vector<F32> lpOutputBuffer_host(this->lpOutputBuffer.size());
		hipMemcpy(&lpOutputBuffer_host[0], thrust::raw_pointer_cast(&this->lpOutputBuffer[0]), sizeof(F32)*lpOutputBuffer_host.size(), hipMemcpyDeviceToHost);
#endif


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER GlobalAveragePooling_GPU::GetOutputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpOutputBuffer[0]);
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode GlobalAveragePooling_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32)*outputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode GlobalAveragePooling_GPU::Training(CONST_BATCH_BUFFER_POINTER i_lppDOutputBufferPrev)
	{
		// �o�͌덷�o�b�t�@�̃A�h���X��z��Ɋi�[
		this->m_lppDOutputBufferPrev = i_lppDOutputBufferPrev;

		// ���͌덷�o�b�t�@��0�N���A
		hipMemset(thrust::raw_pointer_cast(&this->lpDInputBuffer[0]), 0, sizeof(F32)*this->lpDInputBuffer.size());

		// ch���Ŋ������l����
		{
			dim3 grid((this->chSize + (BLOCK_SIZE-1))/BLOCK_SIZE, this->GetInputDataStruct().ch, this->batchSize);

			cuda_func_DOutput_to_DInput<<<grid, BLOCK_SIZE>>>(
				this->m_lppDOutputBufferPrev,
				thrust::raw_pointer_cast(&this->lpDInputBuffer[0]),
				this->chSize);
		}
		
#ifdef _DEBUG
		std::vector<F32> lpTmpDOutputBuffer_host(this->outputBufferCount * this->batchSize);
		std::vector<F32> lpTmpDInputBuffer_host(this->inputBufferCount * this->batchSize);
		hipMemcpy(&lpTmpDOutputBuffer_host[0], this->m_lppDOutputBufferPrev, sizeof(F32)*lpTmpDOutputBuffer_host.size(), hipMemcpyDeviceToHost);
		hipMemcpy(&lpTmpDInputBuffer_host[0],  thrust::raw_pointer_cast(&this->lpDInputBuffer[0]), sizeof(F32)*lpTmpDInputBuffer_host.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER GlobalAveragePooling_GPU::GetDInputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpDInputBuffer[0]);
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode GlobalAveragePooling_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32)*inputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
