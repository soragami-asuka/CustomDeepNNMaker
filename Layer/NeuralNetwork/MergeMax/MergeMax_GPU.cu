#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"MergeMax_DATA.hpp"
#include"MergeMax_FUNC.hpp"
#include"MergeMax_Base.h"

#include"MergeMax_GPU.cuh"
#include"MergeMax_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


#define CALC_BATCH_MAX	(256)
#define CALC_INPUT_MAX	(1024)

	__global__ void device_FillValue(U32 bufferCount, F32 lpOutputBuffer[], F32 value)
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= bufferCount)
			return;

		lpOutputBuffer[batchNum * bufferCount + bufNum] = value;
	}
	__global__ void device_CalculateMax(U32 maxBufferCount, U32 inputBufferCount, U32 outputBufferCount, const F32 lpInputBuffer[], F32 lpOutputBuffer[])
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= maxBufferCount)
			return;

		lpOutputBuffer[batchNum * outputBufferCount + bufNum] = max(lpOutputBuffer[batchNum * outputBufferCount + bufNum], lpInputBuffer[batchNum * inputBufferCount + bufNum]);
	}
	__global__ void device_CalculateDInput(U32 maxBufferCount, U32 inputBufferCount, U32 outputBufferCount, const F32 lpInputBuffer[], const F32 lpOutputBuffer[], F32 lpDInputBuffer[], const F32 lpDOutputBuffer[])
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= maxBufferCount)
			return;

		U32 inputPos  = batchNum * inputBufferCount  + bufNum;
		U32 outputPos = batchNum * outputBufferCount + bufNum;

		lpDInputBuffer[inputPos] = (lpOutputBuffer[outputPos] == lpInputBuffer[inputPos]) * lpDOutputBuffer[outputPos];
	}


	/** �R���X�g���N�^ */
	MergeMax_GPU::MergeMax_GPU(Gravisbell::GUID guid, MergeMax_LayerData_GPU& i_layerData, const std::vector<IODataStruct>& i_lpInputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	MergeMax_Base					(guid, i_lpInputDataStruct, i_layerData.GetOutputDataStruct(&i_lpInputDataStruct[0], (U32)i_lpInputDataStruct.size()))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	MergeMax_GPU::~MergeMax_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 MergeMax_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode MergeMax_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	MergeMax_LayerData_Base& MergeMax_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const MergeMax_LayerData_Base& MergeMax_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMax_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMax_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->lpInputBufferCount.resize(this->GetInputDataCount());
		for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
		{
			this->lpInputBufferCount[inputNum] = this->GetInputBufferCount(inputNum);
			if(this->lpInputBufferCount[inputNum] == 0)
				return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;
		}

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;



		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMax_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode MergeMax_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// �o�̓o�b�t�@��������
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
		{
			dim3 grid(
				(this->outputBufferCount + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
				min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
			dim3 block(
				min(this->outputBufferCount, CALC_INPUT_MAX));

			device_FillValue<<<grid, block>>>(
				this->outputBufferCount,
				o_lppOutputBuffer,
				-FLT_MAX);
		}


#ifdef _DEBUG
		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], o_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
		{
			U32 bufferSize = min(this->lpInputBufferCount[inputNum], this->outputBufferCount);

			for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
			{
				dim3 grid(
					(bufferSize + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
					min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
				dim3 block(
					min(bufferSize, CALC_INPUT_MAX));

				device_CalculateMax<<<grid, block>>>(
					bufferSize,
					this->lpInputBufferCount[inputNum],
					this->outputBufferCount,
					i_lppInputBuffer[inputNum],
					o_lppOutputBuffer);
			}
			hipDeviceSynchronize();
		}


#ifdef _DEBUG
		std::vector<std::vector<float>> lpTmpInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpInputBuffer[i][0], i_lppInputBuffer[i], sizeof(float)*lpTmpInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}

		hipMemcpy(&lpTmpOutputBuffer[0], o_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeMax_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		if(o_lppDInputBuffer)
		{
			// ���͌덷�o�b�t�@�̏�����
			for(U32 inputNum=0; inputNum<this->GetInputDataCount(); inputNum++)
			{
				hipMemset(o_lppDInputBuffer[inputNum], 0, sizeof(F32)*this->lpInputBufferCount[inputNum]*this->GetBatchSize());
			}


			for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
			{
				U32 bufferSize = min(this->lpInputBufferCount[inputNum], this->outputBufferCount);

				for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
				{
					dim3 grid(
						(bufferSize + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
						min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
					dim3 block(
						min(bufferSize, CALC_INPUT_MAX));

					device_CalculateDInput<<<grid, block>>>(
						bufferSize,
						this->lpInputBufferCount[inputNum],
						this->outputBufferCount,
						i_lppInputBuffer[inputNum],
						i_lppOutputBuffer,
						o_lppDInputBuffer[inputNum],
						i_lppDOutputBuffer);
				}

				hipDeviceSynchronize();
			}
		}


#ifdef _DEBUG
		std::vector<std::vector<float>> lpTmpInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpInputBuffer[i][0], i_lppInputBuffer[i], sizeof(float)*lpTmpInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}

		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], i_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<std::vector<float>> lpTmpDInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpDInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpDInputBuffer[i][0], o_lppDInputBuffer[i], sizeof(float)*lpTmpDInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeMax_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}



} // Gravisbell;
} // Layer;
} // NeuralNetwork;
