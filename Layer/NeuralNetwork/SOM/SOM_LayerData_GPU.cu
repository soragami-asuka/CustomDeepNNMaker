//======================================
// �S�����j���[�����l�b�g���[�N�̃��C���[�f�[�^
// GPU����
//======================================
#include"stdafx.h"

#include"SOM_LayerData_GPU.cuh"
#include"SOM_FUNC.hpp"
#include"SOM_GPU.cuh"

#include"../_LayerBase/CLayerBase_GPU.cuh"

#pragma warning(push)
#pragma warning(disable : 4267)
#include <hip/hip_runtime.h> // need CUDA_VERSION
#include <hipDNN.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include ""
#pragma warning(pop)

#include"Library/NeuralNetwork/Optimizer.h"
#include"Library/NeuralNetwork/Initializer.h"


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	//===========================
	// �R���X�g���N�^ / �f�X�g���N�^
	//===========================
	/** �R���X�g���N�^ */
	SOM_LayerData_GPU::SOM_LayerData_GPU(const Gravisbell::GUID& guid)
		:	SOM_LayerData_Base(guid)
	{
	}
	/** �f�X�g���N�^ */
	SOM_LayerData_GPU::~SOM_LayerData_GPU()
	{
	}


	//===========================
	// ������
	//===========================
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode SOM_LayerData_GPU::Initialize(void)
	{
		// ���̓o�b�t�@�����m�F
		unsigned int inputBufferCount = this->GetInputBufferCount();
		if(inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_COMMON_OUT_OF_VALUERANGE;

		// �j���[���������m�F
		unsigned int unitCount = this->GetUnitCount();
		if(unitCount == 0)
			return ErrorCode::ERROR_CODE_COMMON_OUT_OF_VALUERANGE;

		// �o�b�t�@���m�ۂ��A�����l��ݒ�
		U32 inputCount  = inputBufferCount;
		U32 outputCount = this->layerStructure.DimensionCount;

		this->lpUnitData.resize(unitCount * inputBufferCount);

		thrust::host_vector<F32> lpTmpUnitData(unitCount * inputBufferCount);

		for(U32 i=0; i<lpTmpUnitData.size(); i++)
		{
			lpTmpUnitData[i] = Gravisbell::Layer::NeuralNetwork::GetInitializerManager().GetRandomValue(this->layerStructure.InitializeMinValue, this->layerStructure.InitializeMaxValue);
		}

		this->lpUnitData = lpTmpUnitData;

		return ErrorCode::ERROR_CODE_NONE;
	}
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@param	i_config			�ݒ���
		@oaram	i_inputDataStruct	���̓f�[�^�\�����
		@return	���������ꍇ0 */
	ErrorCode SOM_LayerData_GPU::Initialize(const SettingData::Standard::IData& i_data)
	{
		ErrorCode err;

		// �ݒ���̓o�^
		err = this->SetLayerConfig(i_data);
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		// ������
		err = this->Initialize();
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		// �I�v�e�B�}�C�U�[�̐ݒ�
		err = this->ChangeOptimizer(L"SGD");
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		return this->Initialize();
	}
	/** ������. �o�b�t�@����f�[�^��ǂݍ���
		@param i_lpBuffer	�ǂݍ��݃o�b�t�@�̐擪�A�h���X.
		@param i_bufferSize	�ǂݍ��݉\�o�b�t�@�̃T�C�Y.
		@return	���������ꍇ0 */
	ErrorCode SOM_LayerData_GPU::InitializeFromBuffer(const BYTE* i_lpBuffer, U64 i_bufferSize, S64& o_useBufferSize )
	{
		S64 readBufferByte = 0;

		// �ݒ���
		S64 useBufferByte = 0;
		SettingData::Standard::IData* pLayerStructure = CreateLayerStructureSettingFromBuffer(&i_lpBuffer[readBufferByte], i_bufferSize, useBufferByte);
		if(pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_INITLAYER_READ_CONFIG;
		readBufferByte += useBufferByte;
		this->SetLayerConfig(*pLayerStructure);
		delete pLayerStructure;

		// ����������
		this->Initialize();

		// �o�b�t�@����R�s�[
		// �j���[����
		hipMemcpy(
			thrust::raw_pointer_cast(&this->lpUnitData[0]),
			&i_lpBuffer[readBufferByte],
			sizeof(F32) * this->lpUnitData.size(),
			hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32) * (S32)this->lpUnitData.size();


		o_useBufferSize = readBufferByte;

		return ErrorCode::ERROR_CODE_NONE;
	}


	//===========================
	// ���C���[�ۑ�
	//===========================
	/** ���C���[���o�b�t�@�ɏ�������.
		@param o_lpBuffer	�������ݐ�o�b�t�@�̐擪�A�h���X. GetUseBufferByteCount�̖߂�l�̃o�C�g�����K�v
		@return ���������ꍇ�������񂾃o�b�t�@�T�C�Y.���s�����ꍇ�͕��̒l */
	S64 SOM_LayerData_GPU::WriteToBuffer(BYTE* o_lpBuffer)const
	{
		if(this->pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_NONREGIST_CONFIG;

		S64 writeBufferByte = 0;

		// �ݒ���
		writeBufferByte += this->pLayerStructure->WriteToBuffer(&o_lpBuffer[writeBufferByte]);

		// �j���[����
		hipMemcpy(
			&o_lpBuffer[writeBufferByte],
			thrust::raw_pointer_cast(&this->lpUnitData[0]),
			sizeof(F32) * this->lpUnitData.size(),
			hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32) * (S32)this->lpUnitData.size();


		return writeBufferByte;
	}


	//===========================
	// ���C���[�쐬
	//===========================
	/** ���C���[���쐬����.
		@param guid	�V�K�������郌�C���[��GUID. */
	ILayerBase* SOM_LayerData_GPU::CreateLayer(const Gravisbell::GUID& guid, const IODataStruct i_lpInputDataStruct[], U32 i_inputLayerCount, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
	{
		if(this->CheckCanUseInputDataStruct(i_lpInputDataStruct, i_inputLayerCount) == false)
			return NULL;

		return new CNNSingle2SingleLayerBase_GPU<SOM_GPU, SOM_LayerData_GPU>(guid, *this, i_lpInputDataStruct[0], i_temporaryMemoryManager);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;


using namespace Gravisbell;

/** Create a layer for GPU processing.
  * @param GUID of layer to create.
  */
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPU(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const Gravisbell::SettingData::Standard::IData& i_data)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::SOM_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::SOM_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	Gravisbell::ErrorCode errCode = pLayerData->Initialize(i_data);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	return pLayerData;
}
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPUfromBuffer(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const BYTE* i_lpBuffer, S64 i_bufferSize, S64& o_useBufferSize)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::SOM_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::SOM_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	S64 useBufferSize = 0;
	Gravisbell::ErrorCode errCode = pLayerData->InitializeFromBuffer(i_lpBuffer, i_bufferSize, useBufferSize);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	// �g�p�����o�b�t�@�ʂ��i�[
	o_useBufferSize = useBufferSize;

	return pLayerData;
}