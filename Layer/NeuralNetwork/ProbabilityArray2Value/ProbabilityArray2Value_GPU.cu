#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"
#define _USE_MATH_DEFINES
#include<math.h>

#include"ProbabilityArray2Value_DATA.hpp"
#include"ProbabilityArray2Value_FUNC.hpp"
#include"ProbabilityArray2Value_Base.h"

#include"ProbabilityArray2Value_GPU.cuh"
#include"ProbabilityArray2Value_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

#define CALC_BATCH_MAX	(256)
#define CALC_INPUT_MAX	(1024)


	__global__ void device_Value2ProbabilityArray(
		U32 outputBatchBufferSize,
		F32 outputMinValue,
		F32 outputMaxValue,
		F32 variance,
		F32 lpDInputBuffer[],
		const F32 lpOutputBuffer[],
		const F32 lpDOutputBuffer[])
	{
		U32 batchNum  = blockIdx.x;
		U32 bufferPos = threadIdx.x;
		U32 inputCh = threadIdx.y;
		U32 inputChBufferSize = blockDim.x;
		U32 inputChSize = blockDim.y;

		U32 outputOffset = outputBatchBufferSize * batchNum + bufferPos;

		F32 trueValue = lpOutputBuffer[outputOffset] + lpDOutputBuffer[outputOffset];
		F32 value = (F32)inputCh / inputChSize
				  * (outputMaxValue - outputMinValue)
				  + outputMinValue;

		U32 inputOffset = (inputChBufferSize * inputChSize * batchNum) + (inputChBufferSize * inputCh) + bufferPos;

		lpDInputBuffer[inputOffset] = 1.0f / (2.0f * (F32)M_PI * variance) * expf(-(value - trueValue)*(value - trueValue) / (2.0f * variance * variance));
	}


	/** �R���X�g���N�^ */
	ProbabilityArray2Value_GPU::ProbabilityArray2Value_GPU(Gravisbell::GUID guid, ProbabilityArray2Value_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	ProbabilityArray2Value_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
		,	temporaryMemoryManager			(i_temporaryMemoryManager)
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	ProbabilityArray2Value_GPU::~ProbabilityArray2Value_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 ProbabilityArray2Value_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode ProbabilityArray2Value_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	ProbabilityArray2Value_LayerData_Base& ProbabilityArray2Value_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const ProbabilityArray2Value_LayerData_Base& ProbabilityArray2Value_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode ProbabilityArray2Value_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode ProbabilityArray2Value_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// ���͐M���̃`�����l�����Ƃ̃o�b�t�@�T�C�Y
		this->inputChannelSize = this->GetOutputDataStruct().x * this->GetOutputDataStruct().y * this->GetOutputDataStruct().z;

		/**< ���͐M���̃o�b�`���Ƃ̃o�b�t�@�T�C�Y */
		this->inputBatchBufferSize = this->inputChannelSize * this->GetInputDataStruct().ch;

		// �ꎞ�o�̓o�b�t�@(�z�X�g������)
		this->lpTmpOutputBuffer_h.resize(this->outputBufferCount * this->GetBatchSize());
		this->lpTmpBatchOutputBuffer_h.resize(this->GetBatchSize());
		for(U32 i=0; i<this->GetBatchSize(); i++)
			this->lpTmpBatchOutputBuffer_h[i] = &this->lpTmpOutputBuffer_h[this->outputBufferCount * i];

		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode ProbabilityArray2Value_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode ProbabilityArray2Value_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// �o�̓o�b�t�@�̏�����
		hipMemset(o_lppOutputBuffer, 0, sizeof(F32)*this->outputBufferCount*this->GetBatchSize());
		memset(&this->lpTmpOutputBuffer_h[0], 0, sizeof(F32)*this->lpTmpOutputBuffer_h.size());

		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
		{
			for(U32 z=0; z<this->GetInputDataStruct().z; z++)
			{
				for(U32 y=0; y<this->GetInputDataStruct().y; y++)
				{
					for(U32 x=0; x<this->GetInputDataStruct().x; x++)
					{
						U32 offset = this->GetInputDataStruct().POSITION_TO_OFFSET(x, y, z, 0);

						// �ő�l�̔ԍ����擾
						S32 maxPos = -1;
						hipblasIsamax(
							this->cublasHandle,
							this->inputBatchBufferSize,
							&i_lppInputBuffer[this->inputBatchBufferSize*batchNum + offset],
							this->inputChannelSize,
							&maxPos);

						if(maxPos <= 0)
							continue;

						this->lpTmpBatchOutputBuffer_h[batchNum][offset]
							= (F32)(maxPos - 1) / this->GetInputDataStruct().ch
							* (this->layerData.layerStructure.outputMaxValue - this->layerData.layerStructure.outputMinValue)
							+ this->layerData.layerStructure.outputMinValue;
					}
				}
			}
		}

		// CPU > GPU
		hipMemcpy(
			o_lppOutputBuffer,
			&this->lpTmpOutputBuffer_h[0],
			sizeof(F32) * this->lpTmpOutputBuffer_h.size(),
			hipMemcpyHostToDevice);


#if _DEBUG
			std::vector<F32> lpInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpInputBuffer[0], i_lppInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], o_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode ProbabilityArray2Value_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// �o�̓o�b�t�@�̏�����
			hipMemset(o_lppDInputBuffer, 0, sizeof(F32)*this->inputBufferCount*this->GetBatchSize());

#if _DEBUG
			std::vector<F32> lpDOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], i_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			dim3 grid = this->GetBatchSize();
			dim3 block = dim3(this->inputChannelSize, this->GetInputDataStruct().ch);

			// ���K���z���v�Z
			device_Value2ProbabilityArray<<<grid, block>>>(
				this->outputBufferCount,
				this->layerData.layerStructure.outputMinValue,
				this->layerData.layerStructure.outputMaxValue,
				this->layerData.layerStructure.variance,
				o_lppDInputBuffer,
				i_lppOutputBuffer,
				i_lppDOutputBuffer);

#if _DEBUG
			std::vector<F32> lpTmpCalctBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpTmpCalctBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			// ���ω�
			for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
			{
				for(U32 bufferPos=0; bufferPos<this->inputChannelSize; bufferPos++)
				{
					// ���v�����߂�
					F32 sumValue = 0.0f;
					hipblasSasum(
						this->cublasHandle,
						this->GetInputDataStruct().ch,
						&o_lppDInputBuffer[batchNum * this->inputBufferCount + bufferPos],
						this->inputChannelSize,
						&sumValue);

					// ���v�Ŋ���
					F32 alpha = 1.0f / sumValue;
					hipblasSscal(
						this->cublasHandle,
						this->GetInputDataStruct().ch,
						&alpha,
						&o_lppDInputBuffer[batchNum * this->inputBufferCount + bufferPos],
						this->inputChannelSize);
				}
			}


#if _DEBUG
			std::vector<F32> lpTeachBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpTeachBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			// �����Əo�͂Ō덷�����
			F32 alpha = -1;
			hipblasSaxpy(
				this->cublasHandle,
				this->inputBufferCount * this->GetBatchSize(),
				&alpha,
				i_lppInputBuffer,
				1,
				o_lppDInputBuffer,
				1);

#if _DEBUG
			std::vector<F32> lpDInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDInputBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode ProbabilityArray2Value_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
