#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"Value2SignalArray_DATA.hpp"
#include"Value2SignalArray_FUNC.hpp"
#include"Value2SignalArray_Base.h"

#include"Value2SignalArray_GPU.cuh"
#include"Value2SignalArray_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define WORKSPACE_CODE			L"WorkSpace"

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {
	
#define THREAD_PER_BLOCK	32

	__global__ void device_Value2SignalArray(
		const F32 i_lpValue[],
		F32 o_lpSignalArray[],
		U32 i_resolution,
		F32 i_minValue,
		F32 i_maxValue,
		U32 i_loopCount,
		U32 i_bufferPerCh)
	{
		U32 batchNum     = blockIdx.x;
		U32 inputCh      = blockIdx.y;
		U32 inputChCount = gridDim.y;
		U32 tid          = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = loopNum * THREAD_PER_BLOCK + tid;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 inputOffset = batchNum * inputChCount * i_bufferPerCh + inputCh * i_bufferPerCh + bufferPos;
			F32 inputValue  = i_lpValue[inputOffset];

			// �o�̓`�����l���ԍ���float�Ōv�Z
			U32 outputCh = max(0, min(i_resolution-1, (U32)((i_resolution-1) * (inputValue - i_minValue) / (i_maxValue - i_minValue) + 0.5f) ));
			
			U32 outputOffset = batchNum * (inputChCount*i_resolution) * i_bufferPerCh + (inputCh*i_resolution + outputCh) * i_bufferPerCh + bufferPos;
			o_lpSignalArray[outputOffset] = 1.0f;

			// �����l�ɕϊ�
			//U32 iOutputCh = (U32)fOutputCh;
			//F32 t = fOutputCh - iOutputCh;

			//U32 outputOffset0 = batchNum * (inputChCount*resolution) * i_inputChBufferSize + (inputCh*resolution + iOutputCh + 0) * i_inputChBufferSize + bufferPos;
			//U32 outputOffset1 = batchNum * (inputChCount*resolution) * i_inputChBufferSize + (inputCh*resolution + iOutputCh + 1) * i_inputChBufferSize + bufferPos;

			//lpOutputBuffer[outputOffset0] = (1.0f - t);
			//lpOutputBuffer[outputOffset1] = t;
		}
	}


	/** �M���z���l�ɕϊ�����.
		<inputChNo, batchSize> <32>
		*/
	__global__ void device_SignalArray2Value(
		F32* o_lpTeach,
		const F32* i_lpOutput,
		const F32* i_lpDOutput,
		U32 i_resolution, U32 i_bufferPerCh, U32 i_loopCount,
		F32 i_minValue,
		F32 i_maxValue)
	{
		U32 batchNum = blockIdx.y;
		U32 inputChNo = blockIdx.x;
		U32 inputChCount = gridDim.x;
		U32 tid = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = loopNum * THREAD_PER_BLOCK + tid;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 inputOffset = (batchNum * inputChCount + inputChNo) * i_bufferPerCh + bufferPos;

			// �ő�l�����߂�
			U32 maxNum = 0;
			F32 maxValue = -FLT_MAX;
			for(U32 outputNum=0; outputNum<i_resolution; outputNum++)
			{
				U32 outputChNum = inputChNo * i_resolution + outputNum;
				U32 outputOffset = (batchNum * (inputChCount * i_resolution) + outputChNum) * i_bufferPerCh + bufferPos;

				F32 value = i_lpOutput[outputOffset] + i_lpDOutput[outputOffset];

				if(value > maxValue)
				{
					maxNum = outputNum;
					maxValue = value;
				}
			}

			o_lpTeach[inputOffset] = ((F32)maxNum / (i_resolution-1)) * (i_maxValue - i_minValue) + i_minValue;
		}
	}

	/** �R���X�g���N�^ */
	Value2SignalArray_GPU::Value2SignalArray_GPU(Gravisbell::GUID guid, Value2SignalArray_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	Value2SignalArray_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
		,	temporaryMemoryManager			(i_temporaryMemoryManager)
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	Value2SignalArray_GPU::~Value2SignalArray_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 Value2SignalArray_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode Value2SignalArray_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	Value2SignalArray_LayerData_Base& Value2SignalArray_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const Value2SignalArray_LayerData_Base& Value2SignalArray_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode Value2SignalArray_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// Signal -> value�ϊ����s�����߂̏d�ݔz��̍쐬
		std::vector<F32> lpSignal2ValueWeight_h(this->layerData.layerStructure.resolution);
		for(U32 i=0; i<(U32)this->layerData.layerStructure.resolution; i++)
		{
			lpSignal2ValueWeight_h[i] = (this->layerData.layerStructure.inputMaxValue - this->layerData.layerStructure.inputMinValue) * i / this->layerData.layerStructure.resolution - this->layerData.layerStructure.inputMinValue;
		}

		lpSignal2ValueWeight_d.resize(this->layerData.layerStructure.resolution);
		hipMemcpy(thrust::raw_pointer_cast(&this->lpSignal2ValueWeight_d[0]), &lpSignal2ValueWeight_h[0], sizeof(F32)*lpSignal2ValueWeight_h.size(), hipMemcpyHostToDevice);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Value2SignalArray_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// ���͐M���̃`�����l�����Ƃ̃o�b�t�@�T�C�Y
		this->bufferPerChannel = this->GetOutputDataStruct().x * this->GetOutputDataStruct().y * this->GetOutputDataStruct().z;

		/**< ���͐M���̃o�b�`���Ƃ̃o�b�t�@�T�C�Y */
		this->inputBatchBufferSize = this->bufferPerChannel * this->GetInputDataStruct().ch;

		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Value2SignalArray_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode Value2SignalArray_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// �o�̓o�b�t�@�̏�����
		hipMemset(o_lppOutputBuffer, 0, sizeof(F32)*this->outputBufferCount*this->GetBatchSize());

		dim3 grid(this->GetBatchSize(), this->GetInputDataStruct().ch);
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->bufferPerChannel + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		device_Value2SignalArray<<<grid, block>>>(
			i_lppInputBuffer,
			o_lppOutputBuffer,
			this->layerData.layerStructure.resolution,
			this->layerData.layerStructure.inputMinValue,
			this->layerData.layerStructure.inputMaxValue,
			loopCount,
			this->bufferPerChannel);

#if _DEBUG
			std::vector<F32> lpInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpInputBuffer[0], i_lppInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], o_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Value2SignalArray_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
#if _DEBUG
			std::vector<F32> lpOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpOutputBuffer[0], i_lppOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
			
			std::vector<F32> lpDOutputBuffer(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			dim3 grid(this->GetInputDataStruct().ch, this->GetBatchSize());
			dim3 block(THREAD_PER_BLOCK);
			U32 loopCount = (this->bufferPerChannel + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

			device_SignalArray2Value<<<grid, block>>>(
				o_lppDInputBuffer,
				i_lppOutputBuffer,
				i_lppDOutputBuffer,
				this->layerData.layerStructure.resolution,
				this->bufferPerChannel,
				loopCount,
				this->layerData.layerStructure.inputMinValue, this->layerData.layerStructure.inputMaxValue);

#if _DEBUG
			std::vector<F32> lpDOutputBuffer2(this->outputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDOutputBuffer2[0], i_lppDOutputBuffer, sizeof(F32) * this->outputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);

			std::vector<F32> lpTeachBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpTeachBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

			// �����Ɠ��͂Ō덷�����
			F32 alpha = -1;
			hipblasSaxpy(
				this->cublasHandle,
				this->inputBufferCount * this->GetBatchSize(),
				&alpha,
				i_lppInputBuffer,
				1,
				o_lppDInputBuffer,
				1);

#if _DEBUG
			std::vector<F32> lpDInputBuffer(this->inputBufferCount * this->GetBatchSize());
			hipMemcpy(&lpDInputBuffer[0], o_lppDInputBuffer, sizeof(F32) * this->inputBufferCount * this->GetBatchSize(), hipMemcpyDeviceToHost);
#endif

		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Value2SignalArray_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
