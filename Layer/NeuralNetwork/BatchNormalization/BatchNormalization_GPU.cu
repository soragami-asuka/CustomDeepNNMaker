#include "hip/hip_runtime.h"
//======================================
// �o�b�`���K�����C���[
// GPU�����p
//======================================
#include"stdafx.h"

#include"BatchNormalization_DATA.hpp"
#include"BatchNormalization_FUNC.hpp"
#include"BatchNormalization_Base.h"

#include"BatchNormalization_GPU.cuh"
#include"BatchNormalization_LayerData_GPU.cuh"


using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	BatchNormalization_GPU::BatchNormalization_GPU(Gravisbell::GUID guid, BatchNormalization_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct)
		:	BatchNormalization_Base	(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData				(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount		(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount		(0)				/**< �o�̓o�b�t�@�� */
		,	channeclBufferCount		(0)				/**< 1�`�����l��������̃o�b�t�@�� */
		,	onLearnMode				(false)			/**< �w�K�������t���O */
		,	learnCount				(0)				/**< �w�K���s�� */
		,	m_lppInputBuffer				(NULL)			/**< ���Z���̓��̓f�[�^ */
		,	m_lppDOutputBufferPrev			(NULL)			/**< ���͌덷�v�Z���̏o�͌덷�f�[�^ */
	{
        hipdnnCreate(&this->cudnnHandle);
		hipdnnCreateTensorDescriptor(&this->paramTensorDesc);
        hipdnnCreateTensorDescriptor(&this->inputTensorDesc);
        hipdnnCreateTensorDescriptor(&this->outputTensorDesc);
	}
	/** �f�X�g���N�^ */
	BatchNormalization_GPU::~BatchNormalization_GPU()
	{
        hipdnnDestroyTensorDescriptor(this->inputTensorDesc);
        hipdnnDestroyTensorDescriptor(this->outputTensorDesc);
		hipdnnDestroyTensorDescriptor(this->paramTensorDesc);
        hipdnnDestroy(this->cudnnHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 BatchNormalization_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode BatchNormalization_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	BatchNormalization_LayerData_Base& BatchNormalization_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const BatchNormalization_LayerData_Base& BatchNormalization_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode BatchNormalization_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// �w�K�p�̕ϐ����쐬
		this->onLearnMode = true;
		this->learnCount = 0;
		this->lpTmpMean.resize(this->GetInputDataStruct().ch, 0.0f);
		this->lpTmpVariance.resize(this->GetInputDataStruct().ch, 0.0f);

		// �p�����[�^�ω���
		this->lpDBias.resize(this->layerData.lpBias.size());
		this->lpDScale.resize(this->layerData.lpScale.size());

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode BatchNormalization_GPU::PreProcessCalculate()
	{
		hipdnnStatus_t err_cudnn;

		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �`�����l�����Ƃ̃o�b�t�@�����m�F
		this->channeclBufferCount = this->GetInputDataStruct().z * this->GetInputDataStruct().y * this->GetInputDataStruct().x;
		if(this->channeclBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@���쐬
		this->lpOutputBuffer.resize(this->GetBatchSize() * this->outputBufferCount);


		// �������𒲂ׂ�
		S32 dataDim = 1 + 1 + 0;	// �o�b�` + �`�����l�� + ����0
		std::vector<S32> dimInput;			// ���̓f�[�^�\��
		std::vector<S32> dimInputStride;	// ���̓f�[�^�̊e�������Ƃ̃f�[�^��
		std::vector<S32> dimOutput;
		std::vector<S32> dimOutputStride;
		std::vector<S32> dimParam;
		std::vector<S32> dimParamStride;
		if(this->GetInputDataStruct().z > 1)
		{
			dataDim = 1 + 1 + 3;	// �o�b�` + �`�����l�� + ����3

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize();
			dimInput[1] = this->GetInputDataStruct().ch;
			dimInput[2] = this->GetInputDataStruct().z;
			dimInput[3] = this->GetInputDataStruct().y;
			dimInput[4] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[1] = dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[2] = dimInput[3] * dimInput[4];
			dimInputStride[3] = dimInput[4];
			dimInputStride[4] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize();
			dimOutput[1] = this->GetOutputDataStruct().ch;
			dimOutput[2] = this->GetOutputDataStruct().z;
			dimOutput[3] = this->GetOutputDataStruct().y;
			dimOutput[4] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[2] = dimOutput[3] * dimOutput[4];
			dimOutputStride[3] = dimOutput[4];
			dimOutputStride[4] = 1;

			dimParam.resize(dataDim);
			dimParam[0] = 1;
			dimParam[1] = this->GetInputDataStruct().ch;
			dimParam[2] = 1;
			dimParam[3] = 1;
			dimParam[4] = 1;

			dimParamStride.resize(dataDim);
			dimParamStride[0] = dimParam[1] * dimParam[2] * dimParam[3] * dimParam[4];
			dimParamStride[1] = dimParam[2] * dimParam[3] * dimParam[4];
			dimParamStride[2] = dimParam[3] * dimParam[4];
			dimParamStride[3] = dimParam[4];
			dimParamStride[4] = 1;
		}
		else if(this->GetInputDataStruct().y > 1 || this->GetInputDataStruct().x)
		{
			dataDim = 1 + 1 + 2;

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize();
			dimInput[1] = this->GetInputDataStruct().ch;
			dimInput[2] = this->GetInputDataStruct().y;
			dimInput[3] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3];
			dimInputStride[1] = dimInput[2] * dimInput[3];
			dimInputStride[2] = dimInput[3];
			dimInputStride[3] = 1;
			
			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize();
			dimOutput[1] = this->GetOutputDataStruct().ch;
			dimOutput[2] = this->GetOutputDataStruct().y;
			dimOutput[3] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3];
			dimOutputStride[2] = dimOutput[3];
			dimOutputStride[3] = 1;

			dimParam.resize(dataDim);
			dimParam[0] = 1;
			dimParam[1] = this->GetInputDataStruct().ch;
			dimParam[2] = 1;
			dimParam[3] = 1;

			dimParamStride.resize(dataDim);
			dimParamStride[0] = dimParam[1] * dimParam[2] * dimParam[3];
			dimParamStride[1] = dimParam[2] * dimParam[3];
			dimParamStride[2] = dimParam[3];
			dimParamStride[3] = 1;
		}
		else if(this->GetInputDataStruct().x > 1)
		{
			dataDim = 1 + 1 + 1;

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize();
			dimInput[1] = this->GetInputDataStruct().ch;
			dimInput[2] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2];
			dimInputStride[1] = dimInput[2];
			dimInputStride[2] = 1;
			
			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize();
			dimOutput[1] = this->GetOutputDataStruct().ch;
			dimOutput[2] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2];
			dimOutputStride[1] = dimOutput[2];
			dimOutputStride[2] = 1;

			dimParam.resize(dataDim);
			dimParam[0] = 1;
			dimParam[1] = this->GetInputDataStruct().ch;
			dimParam[2] = 1;

			dimParamStride.resize(dataDim);
			dimParamStride[0] = dimParam[1] * dimParam[2];
			dimParamStride[1] = dimParam[2];
			dimParamStride[2] = 1;
		}
		else
		{
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;
		}


		// CUDNN�̓��̓f�[�^�\�����쐬
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->inputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimInput[0],
			&dimInputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// CUDNN�̏o�̓f�[�^�\�����쐬
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->outputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimOutput[0],
			&dimOutputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// CUDNN�̃p�����[�^�f�[�^�\�����쐬
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->paramTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimParam[0],
			&dimParamStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		return ErrorCode::ERROR_CODE_NONE;
	}

	
	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode BatchNormalization_GPU::PreProcessLoop()
	{
		switch(this->GetProcessType())
		{
		case ProcessType::PROCESSTYPE_LEARN:
			{
				// �w�K�񐔂�������
				this->learnCount = 0;

				// ���Z�p�̕���.���U��������
				hipMemset(thrust::raw_pointer_cast(&this->layerData.lpMean[0]),	 0, sizeof(F32)*this->GetInputDataStruct().ch);
				hipMemset(thrust::raw_pointer_cast(&this->layerData.lpVariance[0]), 0, sizeof(F32)*this->GetInputDataStruct().ch);

				hipMemset(thrust::raw_pointer_cast(&this->lpLearnMean[0]),		0, sizeof(F32)*this->GetInputDataStruct().ch);
				hipMemset(thrust::raw_pointer_cast(&this->lpLearnVariance[0]),	0, sizeof(F32)*this->GetInputDataStruct().ch);
			}
			break;
		case ProcessType::PROCESSTYPE_CALCULATE:
			{
				// ����,���U���ꎞ�o�b�t�@�Ɉڂ�
				this->lpTmpMean = this->layerData.lpMean;
				this->lpTmpVariance = this->layerData.lpVariance;

				this->lpLearnMean = this->layerData.lpMean;
				this->lpLearnVariance = this->layerData.lpVariance;
			}
			break;
		}

		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode BatchNormalization_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// ���̓o�b�t�@�̃A�h���X���i�[
		this->m_lppInputBuffer = i_lpInputBuffer;

		// �w�K���Ȃ�Ε��ρA���U�����߂�
		if(this->onLearnMode)
		{
			// �w�K���̏ꍇ
			F32 alpha = 1.0f;
			F32 beta = 0.0f;

			std::vector<F32> lpVarianceLast(this->GetInputDataStruct().ch);
			for(U32 i=0; i<lpVarianceLast.size(); i++)
				lpVarianceLast[i] = this->layerData.lpVariance[i];

			// ���ρA���U���w�K�p�Ɉڂ�
			this->lpLearnMean     = this->layerData.lpMean;
			this->lpLearnVariance = this->layerData.lpVariance;

			err_cudnn = hipdnnBatchNormalizationForwardTraining(
				this->cudnnHandle,
				hipdnnBatchNormMode_t::HIPDNN_BATCHNORM_SPATIAL,
				&alpha,
				&beta,
				this->inputTensorDesc,
				this->m_lppInputBuffer,
				this->outputTensorDesc,
				thrust::raw_pointer_cast(&this->lpOutputBuffer[0]),
				this->paramTensorDesc,
				thrust::raw_pointer_cast(&this->layerData.lpScale[0]),
				thrust::raw_pointer_cast(&this->layerData.lpBias[0]),
				(1.0 / (this->learnCount+1)),
				thrust::raw_pointer_cast(&this->lpLearnMean[0]),
				thrust::raw_pointer_cast(&this->lpLearnVariance[0]),
				max(HIPDNN_BN_MIN_EPSILON, this->layerData.layerStructure.epsilon),
				thrust::raw_pointer_cast(&this->lpTmpMean[0]),
				thrust::raw_pointer_cast(&this->lpTmpVariance[0]));
			if(err_cudnn != 0)
				return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
		}
		else
		{
			// �w�K���łȂ��ꍇ
			F32 alpha = 1.0f;
			F32 beta = 0.0f;

			err_cudnn = hipdnnBatchNormalizationForwardInference(
				this->cudnnHandle,
				hipdnnBatchNormMode_t::HIPDNN_BATCHNORM_SPATIAL,
				&alpha,
				&beta,
				this->inputTensorDesc,
				this->m_lppInputBuffer,
				this->outputTensorDesc,
				thrust::raw_pointer_cast(&this->lpOutputBuffer[0]),
				this->paramTensorDesc,
				thrust::raw_pointer_cast(&this->layerData.lpScale[0]),
				thrust::raw_pointer_cast(&this->layerData.lpBias[0]),
				thrust::raw_pointer_cast(&this->layerData.lpMean[0]),
				thrust::raw_pointer_cast(&this->layerData.lpVariance[0]),
				max(HIPDNN_BN_MIN_EPSILON, this->layerData.layerStructure.epsilon));
			if(err_cudnn != 0)
				return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER BatchNormalization_GPU::GetOutputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpOutputBuffer[0]);
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode BatchNormalization_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		CONST_BATCH_BUFFER_POINTER lppUseOutputBuffer = this->GetOutputBuffer();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32) * outputBufferCount * batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode BatchNormalization_GPU::CalculateDInput(BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// �o�͌덷�o�b�t�@�̃A�h���X���i�[
		this->m_lppDOutputBufferPrev = i_lppDOutputBuffer;

		// ���͌덷�o�b�t�@�̃A�h���X���i�[
		this->m_lpDInputBuffer_d = o_lppDInputBuffer;
		if(this->m_lpDInputBuffer_d == NULL)
		{
			// ���͌덷�o�b�t�@�����݂��Ȃ��ꍇ�w�K���ł��Ȃ����߁A��փo�b�t�@���m��
			if(this->m_lpTemporaryDInputBuffer_d.size() != this->inputBufferCount * this->GetBatchSize())
				this->m_lpTemporaryDInputBuffer_d.resize(this->inputBufferCount * this->GetBatchSize());

			this->m_lpDInputBuffer_d = thrust::raw_pointer_cast(&this->m_lpTemporaryDInputBuffer_d[0]);
		}


		F32 alphaData = 1.0f;
		F32 betaData  = 0.0f;

		F32 alphaParam = 0.0f;
		F32 betaParam  = 1.0f;

		err_cudnn = hipdnnBatchNormalizationBackward(
			this->cudnnHandle,
			hipdnnBatchNormMode_t::HIPDNN_BATCHNORM_SPATIAL,
			&alphaData,
			&betaData,
			&alphaParam,
			&betaParam,
			this->inputTensorDesc,
			this->m_lppInputBuffer,
			this->outputTensorDesc,
			this->m_lppDOutputBufferPrev,
			this->inputTensorDesc,
			this->m_lpDInputBuffer_d,
			this->paramTensorDesc,
			thrust::raw_pointer_cast(&this->layerData.lpScale[0]),
			thrust::raw_pointer_cast(&this->layerData.lpScale[0]),
			thrust::raw_pointer_cast(&this->layerData.lpBias[0]),
			max(HIPDNN_BN_MIN_EPSILON, this->layerData.layerStructure.epsilon),
			thrust::raw_pointer_cast(&this->lpTmpMean[0]),
			thrust::raw_pointer_cast(&this->lpTmpVariance[0]));
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_CALCULATE;

#ifdef _DEBUG
		std::vector<float> lpTmpInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpInputBuffer[0], this->m_lppInputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpInputBuffer[0], thrust::raw_pointer_cast(&this->lpOutputBuffer[0]), sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpDInputBuffer[0], o_lppDInputBuffer, sizeof(float)*lpTmpDInputBuffer.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode BatchNormalization_GPU::Training(BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// �o�͌덷�o�b�t�@�̃A�h���X���i�[
		this->m_lppDOutputBufferPrev = i_lppDOutputBuffer;

		// ���͌덷�o�b�t�@�̃A�h���X���i�[
		this->m_lpDInputBuffer_d = o_lppDInputBuffer;
		if(this->m_lpDInputBuffer_d == NULL)
		{
			// ���͌덷�o�b�t�@�����݂��Ȃ��ꍇ�w�K���ł��Ȃ����߁A��փo�b�t�@���m��
			if(this->m_lpTemporaryDInputBuffer_d.size() != this->inputBufferCount * this->GetBatchSize())
				this->m_lpTemporaryDInputBuffer_d.resize(this->inputBufferCount * this->GetBatchSize());

			this->m_lpDInputBuffer_d = thrust::raw_pointer_cast(&this->m_lpTemporaryDInputBuffer_d[0]);
		}


		F32 alphaData = 1.0f;
		F32 betaData  = 0.0f;

		F32 alphaParam = 1.0F;
		F32 betaParam  = 0.0F;

		err_cudnn = hipdnnBatchNormalizationBackward(
			this->cudnnHandle,
			hipdnnBatchNormMode_t::HIPDNN_BATCHNORM_SPATIAL,
			&alphaData,
			&betaData,
			&alphaParam,
			&betaParam,
			this->inputTensorDesc,
			this->m_lppInputBuffer,
			this->outputTensorDesc,
			this->m_lppDOutputBufferPrev,
			this->inputTensorDesc,
			this->m_lpDInputBuffer_d,
			this->paramTensorDesc,
			thrust::raw_pointer_cast(&this->layerData.lpScale[0]),
			thrust::raw_pointer_cast(&this->lpDScale[0]),
			thrust::raw_pointer_cast(&this->lpDBias[0]),
			max(HIPDNN_BN_MIN_EPSILON, this->layerData.layerStructure.epsilon),
			thrust::raw_pointer_cast(&this->lpTmpMean[0]),
			thrust::raw_pointer_cast(&this->lpTmpVariance[0]));
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_CALCULATE;

		// ���ρA���U���X�V
		this->layerData.lpMean = this->lpLearnMean;
		this->layerData.lpVariance = this->lpLearnVariance;

		// �p�����[�^���X�V
		if(this->layerData.m_pOptimizer_scale)
			this->layerData.m_pOptimizer_scale->UpdateParameter(thrust::raw_pointer_cast(&this->layerData.lpScale[0]), thrust::raw_pointer_cast(&this->lpDScale[0]));
		if(this->layerData.m_pOptimizer_bias)
			this->layerData.m_pOptimizer_bias->UpdateParameter(thrust::raw_pointer_cast(&this->layerData.lpBias[0]), thrust::raw_pointer_cast(&this->lpDBias[0]));

		// �w�K�����̎��s�񐔂��J�E���g�A�b�v
		this->learnCount++;


#ifdef _DEBUG
		std::vector<float> lpMean_h(this->layerData.lpMean.size());
		hipMemcpy(&lpMean_h[0], thrust::raw_pointer_cast(&this->layerData.lpMean[0]), sizeof(float)*this->layerData.lpMean.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpVariance_h(this->layerData.lpVariance.size());
		hipMemcpy(&lpVariance_h[0], thrust::raw_pointer_cast(&this->layerData.lpVariance[0]), sizeof(float)*this->layerData.lpVariance.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpDScale_h(this->lpDBias.size());
		hipMemcpy(&lpDScale_h[0], thrust::raw_pointer_cast(&this->lpDScale[0]), sizeof(float)*lpDScale_h.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpDBias_h(this->lpDBias.size());
		hipMemcpy(&lpDBias_h[0], thrust::raw_pointer_cast(&this->lpDBias[0]), sizeof(float)*lpDBias_h.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpScale_h(this->layerData.lpScale.size());
		hipMemcpy(&lpScale_h[0], thrust::raw_pointer_cast(&this->layerData.lpScale[0]), sizeof(float)*lpScale_h.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpBias_h(this->layerData.lpBias.size());
		hipMemcpy(&lpBias_h[0], thrust::raw_pointer_cast(&this->layerData.lpBias[0]), sizeof(float)*lpBias_h.size(), hipMemcpyDeviceToHost);

#endif

#ifdef _DEBUG
		std::vector<float> lpTmpInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpInputBuffer[0], this->m_lppInputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], thrust::raw_pointer_cast(&this->lpOutputBuffer[0]), sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpDInputBuffer[0], o_lppDInputBuffer, sizeof(float)*lpTmpDInputBuffer.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER BatchNormalization_GPU::GetDInputBuffer()const
	{
		return this->m_lpDInputBuffer_d;
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode BatchNormalization_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		CONST_BATCH_BUFFER_POINTER lppUseDInputBuffer = this->GetDInputBuffer();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32) * inputBufferCount * batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
