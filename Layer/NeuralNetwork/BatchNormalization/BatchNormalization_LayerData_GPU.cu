//======================================
// �o�b�`���K���̃��C���[�f�[�^
// GPU����
//======================================
#include"stdafx.h"

#include"BatchNormalization_LayerData_GPU.cuh"
#include"BatchNormalization_FUNC.hpp"
#include"BatchNormalization_GPU.cuh"

using namespace Gravisbell;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	//===========================
	// �R���X�g���N�^ / �f�X�g���N�^
	//===========================
	/** �R���X�g���N�^ */
	BatchNormalization_LayerData_GPU::BatchNormalization_LayerData_GPU(const Gravisbell::GUID& guid)
		:	BatchNormalization_LayerData_Base(guid)
	{
	}
	/** �f�X�g���N�^ */
	BatchNormalization_LayerData_GPU::~BatchNormalization_LayerData_GPU()
	{
	}


	//===========================
	// ������
	//===========================
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode BatchNormalization_LayerData_GPU::Initialize(void)
	{
		this->lpMean.resize(this->inputDataStruct.ch);
		this->lpVariance.resize(this->inputDataStruct.ch);
		this->lpScale.resize(this->inputDataStruct.ch);
		this->lpBias.resize(this->inputDataStruct.ch);

		for(U32 ch=0; ch<this->inputDataStruct.ch; ch++)
		{
			this->lpMean[ch] = 0.0f;
			this->lpVariance[ch] = 0.0f;
			this->lpScale[ch] = 1.0f;
			this->lpBias[ch] = 0.0f;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@param	i_config			�ݒ���
		@oaram	i_inputDataStruct	���̓f�[�^�\�����
		@return	���������ꍇ0 */
	ErrorCode BatchNormalization_LayerData_GPU::Initialize(const SettingData::Standard::IData& i_data, const IODataStruct& i_inputDataStruct)
	{
		ErrorCode err;

		// �ݒ���̓o�^
		err = this->SetLayerConfig(i_data);
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		// ���̓f�[�^�\���̐ݒ�
		this->inputDataStruct = i_inputDataStruct;

		return this->Initialize();
	}
	/** ������. �o�b�t�@����f�[�^��ǂݍ���
		@param i_lpBuffer	�ǂݍ��݃o�b�t�@�̐擪�A�h���X.
		@param i_bufferSize	�ǂݍ��݉\�o�b�t�@�̃T�C�Y.
		@return	���������ꍇ0 */
	ErrorCode BatchNormalization_LayerData_GPU::InitializeFromBuffer(const BYTE* i_lpBuffer, U32 i_bufferSize, S32& o_useBufferSize )
	{
		int readBufferByte = 0;

		// ���̓f�[�^�\��
		memcpy(&this->inputDataStruct, &i_lpBuffer[readBufferByte], sizeof(this->inputDataStruct));
		readBufferByte += sizeof(this->inputDataStruct);

		// �ݒ���
		S32 useBufferByte = 0;
		SettingData::Standard::IData* pLayerStructure = CreateLayerStructureSettingFromBuffer(&i_lpBuffer[readBufferByte], i_bufferSize, useBufferByte);
		if(pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_INITLAYER_READ_CONFIG;
		readBufferByte += useBufferByte;
		this->SetLayerConfig(*pLayerStructure);
		delete pLayerStructure;

		// ����������
		this->Initialize();

		// ����
		hipMemcpy(thrust::raw_pointer_cast(&this->lpMean[0]), &i_lpBuffer[readBufferByte], sizeof(F32)*this->lpMean.size(), hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32)*(U32)this->lpMean.size();
		// ���U
		hipMemcpy(thrust::raw_pointer_cast(&this->lpVariance[0]), &i_lpBuffer[readBufferByte], sizeof(F32)*this->lpMean.size(), hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32)*(U32)this->lpMean.size();
		// �X�P�[�����O�l
		hipMemcpy(thrust::raw_pointer_cast(&this->lpScale[0]), &i_lpBuffer[readBufferByte], sizeof(F32)*this->lpMean.size(), hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32)*(U32)this->lpMean.size();
		// �o�C�A�X�l
		hipMemcpy(thrust::raw_pointer_cast(&this->lpBias[0]), &i_lpBuffer[readBufferByte], sizeof(F32)*this->lpMean.size(), hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32)*(U32)this->lpMean.size();


		o_useBufferSize = readBufferByte;

		return ErrorCode::ERROR_CODE_NONE;
	}


	//===========================
	// ���C���[�ۑ�
	//===========================
	/** ���C���[���o�b�t�@�ɏ�������.
		@param o_lpBuffer	�������ݐ�o�b�t�@�̐擪�A�h���X. GetUseBufferByteCount�̖߂�l�̃o�C�g�����K�v
		@return ���������ꍇ�������񂾃o�b�t�@�T�C�Y.���s�����ꍇ�͕��̒l */
	S32 BatchNormalization_LayerData_GPU::WriteToBuffer(BYTE* o_lpBuffer)const
	{
		if(this->pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_NONREGIST_CONFIG;

		int writeBufferByte = 0;

		// ���̓f�[�^�\��
		memcpy(&o_lpBuffer[writeBufferByte], &this->inputDataStruct, sizeof(this->inputDataStruct));
		writeBufferByte += sizeof(this->inputDataStruct);

		// �ݒ���
		writeBufferByte += this->pLayerStructure->WriteToBuffer(&o_lpBuffer[writeBufferByte]);

		// ����
		hipMemcpy(&o_lpBuffer[writeBufferByte], thrust::raw_pointer_cast(&this->lpMean[0]), sizeof(F32)*this->lpMean.size(), hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32)*(U32)this->lpMean.size();
		// ���U
		hipMemcpy(&o_lpBuffer[writeBufferByte], thrust::raw_pointer_cast(&this->lpVariance[0]), sizeof(F32)*this->lpVariance.size(), hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32)*(U32)this->lpVariance.size();
		// �X�P�[�����O�l
		hipMemcpy(&o_lpBuffer[writeBufferByte], thrust::raw_pointer_cast(&this->lpScale[0]), sizeof(F32)*this->lpScale.size(), hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32)*(U32)this->lpScale.size();
		// �o�C�A�X�l
		hipMemcpy(&o_lpBuffer[writeBufferByte], thrust::raw_pointer_cast(&this->lpBias[0]), sizeof(F32)*this->lpBias.size(), hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32)*(U32)this->lpBias.size();

		return writeBufferByte;
	}


	//===========================
	// ���C���[�쐬
	//===========================
	/** ���C���[���쐬����.
		@param guid	�V�K�������郌�C���[��GUID. */
	ILayerBase* BatchNormalization_LayerData_GPU::CreateLayer(const Gravisbell::GUID& guid)
	{
		return new BatchNormalization_GPU(guid, *this);
	}

} // Gravisbell;
} // Layer;
} // NeuralNetwork;


/** Create a layer for GPU processing.
  * @param GUID of layer to create.
  */
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPU(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const Gravisbell::SettingData::Standard::IData& i_data, const Gravisbell::IODataStruct& i_inputDataStruct)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::BatchNormalization_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::BatchNormalization_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	Gravisbell::ErrorCode errCode = pLayerData->Initialize(i_data, i_inputDataStruct);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	return pLayerData;
}
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPUfromBuffer(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const BYTE* i_lpBuffer, S32 i_bufferSize, S32& o_useBufferSize)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::BatchNormalization_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::BatchNormalization_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	S32 useBufferSize = 0;
	Gravisbell::ErrorCode errCode = pLayerData->InitializeFromBuffer(i_lpBuffer, i_bufferSize, useBufferSize);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	// �g�p�����o�b�t�@�ʂ��i�[
	o_useBufferSize = useBufferSize;

	return pLayerData;
}
