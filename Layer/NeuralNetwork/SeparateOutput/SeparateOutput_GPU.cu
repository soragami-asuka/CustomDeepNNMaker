//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"SeparateOutput_DATA.hpp"
#include"SeparateOutput_FUNC.hpp"
#include"SeparateOutput_Base.h"

#include"SeparateOutput_GPU.cuh"
#include"SeparateOutput_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	SeparateOutput_GPU::SeparateOutput_GPU(Gravisbell::GUID guid, SeparateOutput_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	SeparateOutput_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	SeparateOutput_GPU::~SeparateOutput_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 SeparateOutput_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode SeparateOutput_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	SeparateOutput_LayerData_Base& SeparateOutput_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const SeparateOutput_LayerData_Base& SeparateOutput_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode SeparateOutput_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode SeparateOutput_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;


		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode SeparateOutput_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode SeparateOutput_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// ���̓o�b�t�@���o�̓o�b�t�@�ɃR�s�[
		hipMemcpy(o_lppOutputBuffer, i_lppInputBuffer, sizeof(F32)*this->outputBufferCount*this->GetBatchSize(), hipMemcpyDeviceToDevice);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode SeparateOutput_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer[])
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// �o�b�t�@1�ڂ̏o�͌덷�ŏ㏑��
			hipMemcpy(o_lppDInputBuffer, i_lppDOutputBuffer[0], sizeof(F32)*this->GetBatchSize()*this->outputBufferCount, hipMemcpyDeviceToDevice);

			// ���Z
			for(U32 outputLayerNum=1; outputLayerNum<(U32)this->layerData.layerStructure.separateCount; outputLayerNum++)
			{
				F32 alpha = 1.0f;

				hipblasSaxpy(
					this->cublasHandle,
					this->GetBatchSize()*this->outputBufferCount,
					&alpha,
					i_lppDOutputBuffer[outputLayerNum], 1,
					o_lppDInputBuffer, 1);
			}
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode SeparateOutput_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer[])
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
