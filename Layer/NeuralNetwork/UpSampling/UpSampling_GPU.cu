//======================================
// ��ݍ��݃j���[�����l�b�g���[�N�̌������C���[
// GPU�����p
//======================================
#include"stdafx.h"

#include"UpSampling_DATA.hpp"
#include"UpSampling_FUNC.hpp"
#include"UpSampling_Base.h"

#include"UpSampling_GPU.cuh"
#include"UpSampling_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	UpSampling_GPU::UpSampling_GPU(Gravisbell::GUID guid, UpSampling_LayerData_GPU& i_layerData)
		:	UpSampling_Base	(guid)
		,	layerData			(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount	(0)		/**< ���̓o�b�t�@�� */
		,	outputBufferCount	(0)		/**< �o�̓o�b�t�@�� */
		,	cudnnHandle			(NULL)
		,	inputTensorDesc		(NULL)
		,	outputTensorDesc	(NULL)
	{
		hipdnnCreate(&cudnnHandle);
		hipdnnCreateTensorDescriptor(&inputTensorDesc);
		hipdnnCreateTensorDescriptor(&upscaleTensorDesc);
		hipdnnCreateTensorDescriptor(&outputTensorDesc);
	}
	/** �f�X�g���N�^ */
	UpSampling_GPU::~UpSampling_GPU()
	{
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(outputTensorDesc);
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(upscaleTensorDesc);
		if(inputTensorDesc)		hipdnnDestroyTensorDescriptor(inputTensorDesc);
		if(cudnnHandle)			hipdnnDestroy(cudnnHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 UpSampling_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode UpSampling_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	UpSampling_LayerData_Base& UpSampling_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const UpSampling_LayerData_Base& UpSampling_GPU::GetLayerData()const
	{
		return this->layerData;
	}



	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessLearn(unsigned int batchSize)
	{
		ErrorCode errorCode = this->PreProcessCalculate(batchSize);
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// ���͍����o�b�t�@���쐬
		this->lpDInputBuffer.resize(this->batchSize * this->inputBufferCount);

		// �j���[����/�o�C�A�X�̌덷���ꎞ�ۑ�����o�b�t�@���쐬
		this->lpDBias.resize(this->layerData.lpBias_d.size());
		this->lppDNeuron.resize(this->layerData.lppNeuron_d.size());

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessCalculate(unsigned int batchSize)
	{
		this->batchSize = batchSize;

		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		hipdnnStatus_t err_cudnn;

		// �������𒲂ׂ�
		S32 dataDim = 1 + 1 + 0;	// �o�b�` + �`�����l�� + ����0
		std::vector<S32> dimInput;			// ���̓f�[�^�\��
		std::vector<S32> dimInputStride;	// ���̓f�[�^�̊e�������Ƃ̃f�[�^��
		std::vector<S32> dimOutput;
		std::vector<S32> dimOutputStride;
		std::vector<S32> dimUpScaleStride;
		if(this->layerData.inputDataStruct.z > 1)
		{
			dataDim = 1 + 1 + 3;

			dimInput.resize(dataDim);
			dimInput[0] = this->batchSize;
			dimInput[1] = this->layerData.inputDataStruct.ch;
			dimInput[2] = this->layerData.inputDataStruct.z;
			dimInput[3] = this->layerData.inputDataStruct.y;
			dimInput[4] = this->layerData.inputDataStruct.x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[1] = dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[2] = dimInput[3] * dimInput[4];
			dimInputStride[3] = dimInput[4];
			dimInputStride[4] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->batchSize;
			dimOutput[1] = this->layerData.outputDataStruct.ch;
			dimOutput[2] = this->layerData.outputDataStruct.z;
			dimOutput[3] = this->layerData.outputDataStruct.y;
			dimOutput[4] = this->layerData.outputDataStruct.x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[2] = dimOutput[3] * dimOutput[4];
			dimOutputStride[3] = dimOutput[4];
			dimOutputStride[4] = 1;

			dimUpScaleStride.resize(dataDim);
			dimUpScaleStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3] * dimOutput[4] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[1] = dimOutput[2] * dimOutput[3] * dimOutput[4] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[2] = dimOutput[3] * dimOutput[4] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[3] = dimOutput[4] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[4] = this->layerData.layerStructure.UpScale.x;

			upscaleStride.x = 1;
			upscaleStride.y = this->layerData.layerStructure.UpScale.x * this->layerData.inputDataStruct.x;
			upscaleStride.z = this->layerData.layerStructure.UpScale.x * this->layerData.inputDataStruct.x * this->layerData.layerStructure.UpScale.y * this->layerData.inputDataStruct.y;
		}
		else if(this->layerData.inputDataStruct.y > 1)
		{
			dataDim = 1 + 1 + 2;

			dimInput.resize(dataDim);
			dimInput[0] = this->batchSize;
			dimInput[1] = this->layerData.inputDataStruct.ch;
			dimInput[2] = this->layerData.inputDataStruct.y;
			dimInput[3] = this->layerData.inputDataStruct.x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3];
			dimInputStride[1] = dimInput[2] * dimInput[3];
			dimInputStride[2] = dimInput[3];
			dimInputStride[3] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->batchSize;
			dimOutput[1] = this->layerData.outputDataStruct.ch;
			dimOutput[2] = this->layerData.outputDataStruct.y;
			dimOutput[3] = this->layerData.outputDataStruct.x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3];
			dimOutputStride[2] = dimOutput[3];
			dimOutputStride[3] = 1;
			
			dimUpScaleStride.resize(dataDim);
			dimUpScaleStride[0] = this->layerData.inputDataStruct.x * this->layerData.layerStructure.UpScale.x * this->layerData.inputDataStruct.y * this->layerData.layerStructure.UpScale.y * this->layerData.inputDataStruct.ch; 
			dimUpScaleStride[1] = this->layerData.inputDataStruct.x * this->layerData.layerStructure.UpScale.x * this->layerData.inputDataStruct.y * this->layerData.layerStructure.UpScale.y;
			dimUpScaleStride[2] = this->layerData.inputDataStruct.x * this->layerData.layerStructure.UpScale.x * this->layerData.layerStructure.UpScale.y;
			dimUpScaleStride[3] = this->layerData.layerStructure.UpScale.x;

			upscaleStride.x = 1;
			upscaleStride.y = this->layerData.layerStructure.UpScale.x * this->layerData.inputDataStruct.x;
			upscaleStride.z = 0;
		}
		else if(this->layerData.inputDataStruct.x > 1)
		{
			dataDim = 1 + 1 + 1;

			dimInput.resize(dataDim);
			dimInput[0] = this->batchSize;
			dimInput[1] = this->layerData.inputDataStruct.ch;
			dimInput[2] = this->layerData.inputDataStruct.x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2];
			dimInputStride[1] = dimInput[2];
			dimInputStride[2] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->batchSize;
			dimOutput[1] = this->layerData.outputDataStruct.ch;
			dimOutput[2] = this->layerData.outputDataStruct.x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2];
			dimOutputStride[1] = dimOutput[2];
			dimOutputStride[2] = 1;

			dimUpScaleStride.resize(dataDim);
			dimUpScaleStride[0] = dimOutput[1] * dimOutput[2] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[1] = dimOutput[2] * this->layerData.layerStructure.UpScale.x;
			dimUpScaleStride[2] = this->layerData.layerStructure.UpScale.x;
			
			upscaleStride.x = 1;
			upscaleStride.y = 0;
			upscaleStride.z = 0;
		}
		else
		{
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;
		}

		// CUDNN�̓��̓f�[�^�\����ݒ�
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->inputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimInput[0],
			&dimInputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_ALLOCATION_MEMORY;


		// CUDNN�̏o�̓f�[�^�\����ݒ�
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->outputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimOutput[0],
			&dimOutputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_ALLOCATION_MEMORY;

		// CUDNN�̓���>�o�͗p�f�[�^�\����ݒ�
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->upscaleTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimInput[0],
			&dimUpScaleStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_ALLOCATION_MEMORY;


		// �o�̓o�b�t�@���쐬
		this->lpOutputBuffer.resize(this->batchSize * this->outputBufferCount);


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K���[�v�̏���������.�f�[�^�Z�b�g�̊w�K�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessLearnLoop(const SettingData::Standard::IData& data)
	{
		if(this->pLearnData != NULL)
			delete this->pLearnData;
		this->pLearnData = data.Clone();

		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}
	/** ���Z���[�v�̏���������.�f�[�^�Z�b�g�̉��Z�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessCalculateLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode UpSampling_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// ���̓o�b�t�@��ۑ�
		this->m_lppInputBuffer_d = i_lpInputBuffer;

		// �o�̓o�b�t�@���N���A
		hipMemset(
			thrust::raw_pointer_cast(&this->lpOutputBuffer[0]),
			0,
			this->lpOutputBuffer.size()*sizeof(F32));

		// ���̓o�b�t�@���o�͂ɃR�s�[
		switch(this->layerData.layerStructure.PaddingType)
		{
		case UpSampling::LayerStructure::PaddingType_value:
			{
				F32 alpha = 1.0f;
				F32 beta  = 1.0f;

				for(S32 offsetZ=0; offsetZ<this->layerData.layerStructure.UpScale.z; offsetZ++)
				{
					for(S32 offsetY=0; offsetY<this->layerData.layerStructure.UpScale.y; offsetY++)
					{
						for(S32 offsetX=0; offsetX<this->layerData.layerStructure.UpScale.x; offsetX++)
						{
							S32 offset = offsetX * this->upscaleStride.x + offsetY * this->upscaleStride.y + offsetZ * this->upscaleStride.z;

							err_cudnn = cudnnTransformTensor(
								this->cudnnHandle,
								&alpha,
								this->inputTensorDesc,
								this->m_lppInputBuffer_d,
								&beta,
								this->upscaleTensorDesc,
								thrust::raw_pointer_cast(&this->lpOutputBuffer[offset]));

							if(err_cudnn != 0)
								return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
						}
					}
				}
			}
			break;
		case UpSampling::LayerStructure::PaddingType_zero:
			{
				F32 alpha = 1.0f;
				F32 beta  = 0.0f;

				err_cudnn = cudnnTransformTensor(
					this->cudnnHandle,
					&alpha,
					this->inputTensorDesc,
					this->m_lppInputBuffer_d,
					&beta,
					this->upscaleTensorDesc,
					thrust::raw_pointer_cast(&this->lpOutputBuffer[0]));

				if(err_cudnn != 0)
					return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
			}
			break;
		}

#ifdef _DEBUG
		std::vector<F32> lpDebugInputBuffer(this->batchSize * this->inputBufferCount);
		hipMemcpy(&lpDebugInputBuffer[0], this->m_lppInputBuffer_d, sizeof(F32)*lpDebugInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<F32> lpDebugOutputBuffer(this->lpOutputBuffer.size());
		hipMemcpy(&lpDebugOutputBuffer[0], thrust::raw_pointer_cast(&this->lpOutputBuffer[0]), sizeof(F32)*lpDebugOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER UpSampling_GPU::GetOutputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpOutputBuffer[0]);
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode UpSampling_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32)*outputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode UpSampling_GPU::Training(CONST_BATCH_BUFFER_POINTER i_lpDOutputBufferPrev)
	{
		hipdnnStatus_t err_cudnn;

		// �o�͌덷�o�b�t�@�̃A�h���X���i�[
		this->m_lppDOutputBuffer_d = i_lpDOutputBufferPrev;

		// ���͌덷�o�b�t�@�̃N���A
		hipMemset(
			thrust::raw_pointer_cast(&this->lpDInputBuffer[0]),
			0,
			this->lpDInputBuffer.size()*sizeof(F32));

		// �o�͌덷�o�b�t�@����͌덷�ɃR�s�[
		switch(this->layerData.layerStructure.PaddingType)
		{
		case UpSampling::LayerStructure::PaddingType_value:
			{
				F32 alpha = 1.0f / (this->layerData.layerStructure.UpScale.x * this->layerData.layerStructure.UpScale.y * this->layerData.layerStructure.UpScale.z);
				F32 beta  = 1.0f;

				for(S32 offsetZ=0; offsetZ<this->layerData.layerStructure.UpScale.z; offsetZ++)
				{
					for(S32 offsetY=0; offsetY<this->layerData.layerStructure.UpScale.y; offsetY++)
					{
						for(S32 offsetX=0; offsetX<this->layerData.layerStructure.UpScale.x; offsetX++)
						{
							S32 offset = offsetX * this->upscaleStride.x + offsetY * this->upscaleStride.y + offsetZ * this->upscaleStride.z;

//#ifdef _DEBUG
//							std::vector<F32> lpPreDebugDInputBuffer(this->lpDInputBuffer.size());
//							hipMemcpy(&lpPreDebugDInputBuffer[0], thrust::raw_pointer_cast(&this->lpDInputBuffer[0]), sizeof(F32)*lpPreDebugDInputBuffer.size(), hipMemcpyDeviceToHost);
//#endif

							err_cudnn = cudnnTransformTensor(
								this->cudnnHandle,
								&alpha,
								this->upscaleTensorDesc,
								&this->m_lppDOutputBuffer_d[offset],
								&beta,
								this->inputTensorDesc,
								thrust::raw_pointer_cast(&this->lpDInputBuffer[0]));

//#ifdef _DEBUG
//							std::vector<F32> lpDebugDOutputBuffer(this->lpOutputBuffer.size());
//							hipMemcpy(&lpDebugDOutputBuffer[0], this->m_lppDOutputBuffer_d, sizeof(F32)*lpDebugDOutputBuffer.size(), hipMemcpyDeviceToHost);
//		
//							std::vector<F32> lpDebugDInputBuffer(this->lpDInputBuffer.size());
//							hipMemcpy(&lpDebugDInputBuffer[0], thrust::raw_pointer_cast(&this->lpDInputBuffer[0]), sizeof(F32)*lpDebugDInputBuffer.size(), hipMemcpyDeviceToHost);
//#endif

							if(err_cudnn != 0)
								return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
							
						}
					}
				}
			}
			break;
		case UpSampling::LayerStructure::PaddingType_zero:
			{
				F32 alpha = 1.0f;
				F32 beta  = 0.0f;

				err_cudnn = cudnnTransformTensor(
					this->cudnnHandle,
					&alpha,
					this->upscaleTensorDesc,
					this->m_lppDOutputBuffer_d,
					&beta,
					this->inputTensorDesc,
					thrust::raw_pointer_cast(&this->lpDInputBuffer[0]));

				if(err_cudnn != 0)
					return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
			}
			break;
		}

#ifdef _DEBUG
		std::vector<F32> lpDebugDOutputBuffer(this->lpOutputBuffer.size());
		hipMemcpy(&lpDebugDOutputBuffer[0], this->m_lppDOutputBuffer_d, sizeof(F32)*lpDebugDOutputBuffer.size(), hipMemcpyDeviceToHost);
		
		std::vector<F32> lpDebugDInputBuffer(this->lpDInputBuffer.size());
		hipMemcpy(&lpDebugDInputBuffer[0], thrust::raw_pointer_cast(&this->lpDInputBuffer[0]), sizeof(F32)*lpDebugDInputBuffer.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER UpSampling_GPU::GetDInputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpDInputBuffer[0]);
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode UpSampling_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32)*inputBufferCount*this->batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
