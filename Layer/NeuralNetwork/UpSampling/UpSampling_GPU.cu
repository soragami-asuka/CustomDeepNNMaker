#include "hip/hip_runtime.h"
//======================================
// ��ݍ��݃j���[�����l�b�g���[�N�̌������C���[
// GPU�����p
//======================================
#include"stdafx.h"

#include"UpSampling_DATA.hpp"
#include"UpSampling_FUNC.hpp"
#include"UpSampling_Base.h"

#include"UpSampling_GPU.cuh"
#include"UpSampling_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	UpSampling_GPU::UpSampling_GPU(Gravisbell::GUID guid, UpSampling_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	UpSampling_Base		(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData			(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount	(0)		/**< ���̓o�b�t�@�� */
		,	outputBufferCount	(0)		/**< �o�̓o�b�t�@�� */
		,	cudnnHandle			(NULL)
		,	inputTensorDesc		(NULL)
		,	outputTensorDesc	(NULL)
		,	filterDesc			(NULL)
		,	convDesc			(NULL)
	{
		hipdnnCreate(&cudnnHandle);
		hipdnnCreateTensorDescriptor(&inputTensorDesc);
		hipdnnCreateTensorDescriptor(&outputTensorDesc);
		hipdnnCreateFilterDescriptor(&filterDesc);
		hipdnnCreateConvolutionDescriptor(&convDesc);
	}
	/** �f�X�g���N�^ */
	UpSampling_GPU::~UpSampling_GPU()
	{
		if(convDesc)			hipdnnDestroyConvolutionDescriptor(convDesc);
		if(filterDesc)			hipdnnDestroyFilterDescriptor(filterDesc);
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(outputTensorDesc);
		if(inputTensorDesc)		hipdnnDestroyTensorDescriptor(inputTensorDesc);
		if(cudnnHandle)			hipdnnDestroy(cudnnHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 UpSampling_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode UpSampling_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	UpSampling_LayerData_Base& UpSampling_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const UpSampling_LayerData_Base& UpSampling_GPU::GetLayerData()const
	{
		return this->layerData;
	}



	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		hipdnnStatus_t err_cudnn;

		// �������𒲂ׂ�
		S32 dataDim = 1 + 1 + 0;	// �o�b�` + �`�����l�� + ����0
		std::vector<S32> dimInput;			// ���̓f�[�^�\��
		std::vector<S32> dimInputStride;	// ���̓f�[�^�̊e�������Ƃ̃f�[�^��
		std::vector<S32> dimOutput;
		std::vector<S32> dimOutputStride;
		S32 filterDim = 0;			// �t�B���^������	���̓`�����l�� + �o�̓`�����l�� + ����
		std::vector<S32> dimFilter;
		S32 convDim = 0;			// ��ݍ��ݎ�����	����
		std::vector<S32> dimStride;
		std::vector<S32> dimDilation;
		std::vector<S32> dimPadding;
		if(this->GetInputDataStruct().z > 1)
		{
			dataDim = 1 + 1 + 3;

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize();
			dimInput[1] = this->GetInputDataStruct().ch;
			dimInput[2] = this->GetInputDataStruct().z;
			dimInput[3] = this->GetInputDataStruct().y;
			dimInput[4] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[1] = dimInput[2] * dimInput[3] * dimInput[4];
			dimInputStride[2] = dimInput[3] * dimInput[4];
			dimInputStride[3] = dimInput[4];
			dimInputStride[4] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize();
			dimOutput[1] = this->GetOutputDataStruct().ch;
			dimOutput[2] = this->GetOutputDataStruct().z;
			dimOutput[3] = this->GetOutputDataStruct().y;
			dimOutput[4] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3] * dimOutput[4];
			dimOutputStride[2] = dimOutput[3] * dimOutput[4];
			dimOutputStride[3] = dimOutput[4];
			dimOutputStride[4] = 1;

			filterDim = 1 + 1 + 2;	// ���̓`�����l�� + �o�̓`�����l�� + ����3

			dimFilter.resize(filterDim);
			dimFilter[0] = this->GetOutputDataStruct().ch;
			dimFilter[1] = this->GetInputDataStruct().ch;
			dimFilter[2] = this->layerData.layerStructure.UpScale.y;
			dimFilter[3] = this->layerData.layerStructure.UpScale.x;

			convDim = 2;	// ����3

			dimPadding.resize(convDim);
			dimPadding[0] = 0;
			dimPadding[1] = 0;

			dimDilation.resize(convDim);
			dimDilation[0] = 1;
			dimDilation[1] = 1;

			dimStride.resize(convDim);
			dimStride[0] = 1;
			dimStride[1] = 1;

		}
		else if(this->GetInputDataStruct().y > 1)
		{
			dataDim = 1 + 1 + 2;

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize() * this->GetInputDataStruct().ch;
			dimInput[1] = 1;
			dimInput[2] = this->GetInputDataStruct().y;
			dimInput[3] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2] * dimInput[3];
			dimInputStride[1] = dimInput[2] * dimInput[3];
			dimInputStride[2] = dimInput[3];
			dimInputStride[3] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize() * this->GetOutputDataStruct().ch;
			dimOutput[1] = 1;
			dimOutput[2] = this->GetOutputDataStruct().y;
			dimOutput[3] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2] * dimOutput[3];
			dimOutputStride[1] = dimOutput[2] * dimOutput[3];
			dimOutputStride[2] = dimOutput[3];
			dimOutputStride[3] = 1;

			filterDim = 1 + 1 + 2;	// ���̓`�����l�� + �o�̓`�����l�� + ����3

			dimFilter.resize(filterDim);
			dimFilter[0] = 1;
			dimFilter[1] = 1;
			dimFilter[2] = this->layerData.layerStructure.UpScale.y;
			dimFilter[3] = this->layerData.layerStructure.UpScale.x;

			convDim = 2;	// ����2

			dimPadding.resize(convDim);
			dimPadding[0] = 0;
			dimPadding[1] = 0;

			dimDilation.resize(convDim);
			dimDilation[0] = 1;
			dimDilation[1] = 1;

			dimStride.resize(convDim);
			dimStride[0] = this->layerData.layerStructure.UpScale.y;
			dimStride[1] = this->layerData.layerStructure.UpScale.x;
		}
		else if(this->GetInputDataStruct().x > 1)
		{
			dataDim = 1 + 1 + 1;

			dimInput.resize(dataDim);
			dimInput[0] = this->GetBatchSize();
			dimInput[1] = this->GetInputDataStruct().ch;
			dimInput[2] = this->GetInputDataStruct().x;

			dimInputStride.resize(dataDim);
			dimInputStride[0] = dimInput[1] * dimInput[2];
			dimInputStride[1] = dimInput[2];
			dimInputStride[2] = 1;

			dimOutput.resize(dataDim);
			dimOutput[0] = this->GetBatchSize();
			dimOutput[1] = this->GetOutputDataStruct().ch;
			dimOutput[2] = this->GetOutputDataStruct().x;

			dimOutputStride.resize(dataDim);
			dimOutputStride[0] = dimOutput[1] * dimOutput[2];
			dimOutputStride[1] = dimOutput[2];
			dimOutputStride[2] = 1;

		}
		else
		{
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;
		}

		// CUDNN�̓��̓f�[�^�\����ݒ�
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->inputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimInput[0],
			&dimInputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_ALLOCATION_MEMORY;

		// CUDNN�̏o�̓f�[�^�\����ݒ�
		err_cudnn = hipdnnSetTensorNdDescriptor(
			this->outputTensorDesc,
			HIPDNN_DATA_FLOAT,
			dataDim,
			&dimOutput[0],
			&dimOutputStride[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_ALLOCATION_MEMORY;

		// �t�B���^�T�C�Y��ݒ�
		err_cudnn = hipdnnSetFilterNdDescriptor(
			this->filterDesc,
			HIPDNN_DATA_FLOAT,
			HIPDNN_TENSOR_NCHW,
			filterDim,
			&dimFilter[0]);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// ��ݍ��ݏ����ݒ�
		err_cudnn = hipdnnSetConvolutionNdDescriptor(
			this->convDesc,
			convDim,
			&dimPadding[0],
			&dimStride[0],
			&dimDilation[0],
			HIPDNN_CROSS_CORRELATION,
			HIPDNN_DATA_FLOAT);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// �ő��̃A���S���Y������������(�O���`�d)
		err_cudnn = hipdnnGetConvolutionForwardAlgorithm(
			this->cudnnHandle,
			this->outputTensorDesc,
			this->filterDesc,
			this->convDesc,
			this->inputTensorDesc,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,	// �������̎g�p�ʖ������ōő��̃A���S���Y���𒲂ׂ�
			0,										// �g�p�\�ȃ������̏��
			&this->useForwardAlgorithm
			);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// �K�v�ȃ������ʂ𒲂ׂ�(�O���`�d)
		size_t workSpaceSizeByte_forward;
		err_cudnn = hipdnnGetConvolutionForwardWorkspaceSize(
			this->cudnnHandle,
			this->outputTensorDesc,
			this->filterDesc,
			this->convDesc,
			this->inputTensorDesc,
			this->useForwardAlgorithm,
			&workSpaceSizeByte_forward);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;


		// �ő��̃A���S���Y������������(����`�d-�f�[�^)
		err_cudnn = hipdnnGetConvolutionBackwardDataAlgorithm(
			this->cudnnHandle,
			this->filterDesc,
			this->inputTensorDesc,
			this->convDesc,
			this->outputTensorDesc,
			hipdnnConvolutionBwdDataPreference_t::HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,	// �������̎g�p�ʖ������ōő��̃A���S���Y���𒲂ׂ�
			0,																				// �g�p�\�ȃ������̏��
			&this->useBackwardDataAlgorithm);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

		// �K�v�ȃ������ʂ𒲂ׂ�(����`�d-�f�[�^)
		size_t workSpaceSizeByte_backwardData;
		err_cudnn = hipdnnGetConvolutionBackwardDataWorkspaceSize(
			this->cudnnHandle,
			this->filterDesc,
			this->inputTensorDesc,
			this->convDesc,
			this->outputTensorDesc,
			this->useBackwardDataAlgorithm,
			&workSpaceSizeByte_backwardData);
		if(err_cudnn != 0)
			return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;


		// �����p�o�b�t�@�̊m��
		this->workSpace.resize(max(workSpaceSizeByte_forward, workSpaceSizeByte_backwardData));


		// �t�B���^�o�b�t�@���쐬���ď�����
		filter.resize(
			this->layerData.layerStructure.UpScale.x * this->layerData.layerStructure.UpScale.y * this->layerData.layerStructure.UpScale.z,
			0.0f);
		for(U32 z=0; z<this->layerData.layerStructure.UpScale.z; z++)
		{
			U32 zOffset = z * this->layerData.layerStructure.UpScale.y * this->layerData.layerStructure.UpScale.x;

			for(U32 y=0; y<this->layerData.layerStructure.UpScale.y; y++)
			{
				U32 yOffset = y * this->layerData.layerStructure.UpScale.x;

				for(U32 x=0; x<this->layerData.layerStructure.UpScale.x; x++)
				{
					U32 offset = zOffset + yOffset + x;

					switch(this->layerData.layerStructure.PaddingType)
					{
					case UpSampling::LayerStructure::PaddingType_value:
						{
							filter[offset] = 1.0f;
						}
						break;
					case UpSampling::LayerStructure::PaddingType_zero:
						{
							if(z==0 && y==0 && x==0)
								filter[offset] = 1.0f;
							else
								filter[offset] = 0.0f;
						}
						break;
					}
				}
			}
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode UpSampling_GPU::PreProcessLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode UpSampling_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// �o�̓o�b�t�@���N���A
		hipMemset(
			o_lppOutputBuffer,
			0,
			this->outputBufferCount * this->GetBatchSize() * sizeof(F32));

		// ���̓o�b�t�@���o�͂ɃR�s�[
		{
			F32 alpha = 1.0f;
			F32 beta  = 0.0f;

			err_cudnn = hipdnnConvolutionBackwardData(
				this->cudnnHandle,
				&alpha,
				this->filterDesc,
				thrust::raw_pointer_cast(&this->filter[0]),
				this->inputTensorDesc,
				i_lppInputBuffer,
				this->convDesc,
				this->useBackwardDataAlgorithm,
				thrust::raw_pointer_cast(&this->workSpace[0]),
				this->workSpace.size(),
				&beta,
				this->outputTensorDesc,
				o_lppOutputBuffer);
			if(err_cudnn != 0)
				return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
		}

#ifdef _DEBUG
		std::vector<F32> lpDebugInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpDebugInputBuffer[0], i_lppInputBuffer, sizeof(F32)*lpDebugInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<F32> lpDebugOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpDebugOutputBuffer[0], o_lppOutputBuffer, sizeof(F32)*lpDebugOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode UpSampling_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		hipdnnStatus_t err_cudnn;

		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// ���͌덷�o�b�t�@�̃N���A
			hipMemset(
				o_lppDInputBuffer,
				0,
				sizeof(F32)*this->inputBufferCount*this->GetBatchSize());

			{
				F32 alpha = 1.0f;
				F32 beta  = 0.0f;
				err_cudnn = hipdnnConvolutionForward(
					this->cudnnHandle,
					&alpha,
					this->outputTensorDesc,
					i_lppDOutputBuffer,
					this->filterDesc,
					thrust::raw_pointer_cast(&this->filter[0]),
					this->convDesc,
					this->useForwardAlgorithm,
					thrust::raw_pointer_cast(&this->workSpace[0]),
					this->workSpace.size(),
					&beta,
					this->inputTensorDesc,
					o_lppDInputBuffer);
				if(err_cudnn != 0)
					return ErrorCode::ERROR_CODE_CUDA_CALCULATE;
			}

		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode UpSampling_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
