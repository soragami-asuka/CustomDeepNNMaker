#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"MergeMultiply_DATA.hpp"
#include"MergeMultiply_FUNC.hpp"
#include"MergeMultiply_Base.h"

#include"MergeMultiply_GPU.cuh"
#include"MergeMultiply_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define Ver01
//#define Ver02

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

#if defined(Ver01)
#define CALC_BATCH_MAX	(256)
#define CALC_INPUT_MAX	(1024)


	__global__ void device_FillValue(U32 bufferCount, F32 lpOutputBuffer[], F32 value)
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= bufferCount)
			return;

		lpOutputBuffer[batchNum * bufferCount + bufNum] = value;
	}
	__global__ void device_CalculateOutput(U32 maxBufferCount, U32 inputBufferCount, U32 outputBufferCount, const F32 lpInputBuffer[], F32 lpOutputBuffer[])
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= maxBufferCount)
			return;

		lpOutputBuffer[batchNum * outputBufferCount + bufNum] *= lpInputBuffer[batchNum * inputBufferCount + bufNum];
	}
	__global__ void device_CalculateDInput(U32 maxBufferCount, U32 inputBufferCount, U32 outputBufferCount, const F32 lpInputBuffer[], const F32 lpOutputBuffer[], F32 lpDInputBuffer[], const F32 lpDOutputBuffer[])
	{
		U32 batchNum = blockIdx.y;
		U32 bufNum   = blockIdx.x * blockDim.x + threadIdx.x;

		if(bufNum >= maxBufferCount)
			return;

		U32 inputPos  = batchNum * inputBufferCount  + bufNum;
		U32 outputPos = batchNum * outputBufferCount + bufNum;

		lpDInputBuffer[inputPos] = (abs(lpOutputBuffer[outputPos]) > 1e-30) ? lpDOutputBuffer[outputPos] * lpOutputBuffer[outputPos] / lpInputBuffer[inputPos] : 0.0f;
	}

#else
	
#define THREAD_PER_BLOCK	32

	/** ���͂𑫂����킹��.
		<outputChCount, batchSize> <32>
		@param	o_lpOutput			�o�̓o�b�t�@
		@param	i_outputChCount		�o�̓o�b�t�@��CH��
		@param	i_inputLyaerCount	���̓��C���[��
		@param	i_lppInput			���̓o�b�t�@
		@param	i_lpInputChCount	���̓o�b�t�@��CH��
		@param	i_bufferPerCh		�`�����l��������̃o�b�t�@��
		@param	i_loopCount			1�X���b�h������̎��s���[�v��
		*/
	__global__ void device_Calculate(F32* o_lpOutput, U32 i_outputChCount, U32 i_inputLayerCount, const F32*const* i_lppInput, const U32* i_lpInputChCount, U32 i_bufferPerCh, U32 i_loopCount)
	{
		U32 chNum    = blockIdx.x;
		U32 batchNum = blockIdx.y;
		U32 tid = threadIdx.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = tid*i_loopCount + loopNum;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 outputOffset = (batchNum * i_outputChCount + chNum) * i_bufferPerCh + bufferPos;

			// �o�͏�����
			o_lpOutput[outputOffset] = 1.0f;
			for(U32 inputLayerNum=0; inputLayerNum<i_inputLayerCount; inputLayerNum++)
			{
				if(chNum >= i_lpInputChCount[inputLayerNum])
					continue;

				U32 inputOffset = (batchNum * i_lpInputChCount[inputLayerNum] + chNum) *i_bufferPerCh + bufferPos;

				o_lpOutput[outputOffset] *= i_lppInput[inputLayerNum][inputOffset];
			}
		}
	}

	/** ���͌덷���v�Z����.
		<outputChCount, batchSize> <32>
		@param	o_lppDInput			���͌덷�o�b�t�@
		@param	i_lpInputChCount	���̓o�b�t�@��CH��
		@param	i_inputLyaerCount	���̓��C���[��
		@param	i_lpDOutput			�o�͌덷�o�b�t�@
		@param	i_bufferPerCh		�`�����l��������̃o�b�t�@��
		@param	i_loopCount			1�X���b�h������̎��s���[�v��
		*/
	__global__ void device_CalculateDInput(F32** o_lppDInput, const F32*const* i_lppInput, const U32* i_lpInputChCount, U32 i_inputLayerCount, const F32* i_lpDOutput, const F32* i_lpOutput, U32 i_bufferPerCh, U32 i_loopCount)
	{
		U32 chNum    = blockIdx.x;
		U32 batchNum = blockIdx.y;
		U32 tid = threadIdx.x;
		U32 outputChCount = gridDim.x;

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = tid*i_loopCount + loopNum;
			if(bufferPos >= i_bufferPerCh)
				continue;

			U32 outputOffset = (batchNum * outputChCount + chNum) * i_bufferPerCh + bufferPos;

			// ���͌덷�v�Z
			for(U32 inputLayerNum=0; inputLayerNum<i_inputLayerCount; inputLayerNum++)
			{
				if(chNum >= i_lpInputChCount[inputLayerNum])
					continue;

				U32 inputOffset = (batchNum * i_lpInputChCount[inputLayerNum] + chNum) *i_bufferPerCh + bufferPos;

				o_lppDInput[inputLayerNum][inputOffset] = abs(i_lpOutput[outputOffset])>0 ? i_lpOutput[outputOffset] / i_lppInput[inputLayerNum][inputOffset] * i_lpDOutput[outputOffset] : 0.0f;
			}
		}
	}

#endif

	/** �R���X�g���N�^ */
	MergeMultiply_GPU::MergeMultiply_GPU(Gravisbell::GUID guid, MergeMultiply_LayerData_GPU& i_layerData, const std::vector<IODataStruct>& i_lpInputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	MergeMultiply_Base					(guid, i_lpInputDataStruct, i_layerData.GetOutputDataStruct(&i_lpInputDataStruct[0], (U32)i_lpInputDataStruct.size()))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	MergeMultiply_GPU::~MergeMultiply_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 MergeMultiply_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode MergeMultiply_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	MergeMultiply_LayerData_Base& MergeMultiply_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const MergeMultiply_LayerData_Base& MergeMultiply_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMultiply_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMultiply_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->lpInputBufferCount.resize(this->GetInputDataCount());
		for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
		{
			this->lpInputBufferCount[inputNum] = this->GetInputBufferCount(inputNum);
			if(this->lpInputBufferCount[inputNum] == 0)
				return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;
		}

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;


		// CH������̃o�b�t�@��
		this->bufferCountPerCh = this->GetOutputDataStruct().x * this->GetOutputDataStruct().y * this->GetOutputDataStruct().z;

		// �e���̓��C���[��CH��
		thrust::host_vector<U32> lpInputChCount(this->GetInputDataCount());
		for(U32 inputNum=0; inputNum<this->GetInputDataCount(); inputNum++)
		{
			lpInputChCount[inputNum] = this->GetInputDataStruct(inputNum).ch;
		}
		this->lpInputChCount_d = lpInputChCount;
		
		// ���͐M���̐擪�A�h���X�̔z��
		// �o�b�t�@�̊m�ۂ̂�
		this->lppInputBuffer_d.resize(this->GetInputDataCount());
		this->lppDInputBuffer_d.resize(this->GetInputDataCount());

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeMultiply_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode MergeMultiply_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
#if defined(Ver01)
		// �o�̓o�b�t�@��������
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
		{
			dim3 grid(
				(this->outputBufferCount + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
				min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
			dim3 block(
				min(this->outputBufferCount, CALC_INPUT_MAX));

			device_FillValue<<<grid, block>>>(
				this->outputBufferCount,
				o_lppOutputBuffer,
				1.0f);
		}


#ifdef _DEBUG
		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], o_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
		{
			U32 bufferSize = min(this->lpInputBufferCount[inputNum], this->outputBufferCount);

			for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
			{
				dim3 grid(
					(bufferSize + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
					min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
				dim3 block(
					min(bufferSize, CALC_INPUT_MAX));

				device_CalculateOutput<<<grid, block>>>(
					bufferSize,
					this->lpInputBufferCount[inputNum],
					this->outputBufferCount,
					i_lppInputBuffer[inputNum],
					o_lppOutputBuffer);
			}
			hipDeviceSynchronize();
		}
#else
		// ���͐M���z���Device�ɃR�s�[
		hipMemcpy(thrust::raw_pointer_cast(&this->lppInputBuffer_d[0]), i_lppInputBuffer, sizeof(F32*)*this->lppInputBuffer_d.size(), hipMemcpyHostToDevice);

		// �v�Z
		dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->bufferCountPerCh + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		device_Calculate<<<grid,block>>>(
			o_lppOutputBuffer, this->GetOutputDataStruct().ch,
			this->GetInputDataCount(), thrust::raw_pointer_cast(&this->lppInputBuffer_d[0]), thrust::raw_pointer_cast(&this->lpInputChCount_d[0]),
			this->bufferCountPerCh,
			loopCount);
#endif



#ifdef _DEBUG
		std::vector<std::vector<float>> lpTmpInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpInputBuffer[i][0], i_lppInputBuffer[i], sizeof(float)*lpTmpInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}

		hipMemcpy(&lpTmpOutputBuffer[0], o_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif


		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeMultiply_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
#if defined(Ver01)
		if(o_lppDInputBuffer)
		{
			// ���͌덷�o�b�t�@�̏�����
			for(U32 inputNum=0; inputNum<this->GetInputDataCount(); inputNum++)
			{
				hipMemset(o_lppDInputBuffer[inputNum], 0, sizeof(F32)*this->lpInputBufferCount[inputNum]*this->GetBatchSize());
			}


			for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
			{
				U32 bufferSize = min(this->lpInputBufferCount[inputNum], this->outputBufferCount);

				for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum+=CALC_BATCH_MAX)
				{
					dim3 grid(
						(bufferSize + (CALC_INPUT_MAX-1))/CALC_INPUT_MAX,
						min(this->GetBatchSize()-batchNum, CALC_BATCH_MAX));
					dim3 block(
						min(bufferSize, CALC_INPUT_MAX));

					device_CalculateDInput<<<grid, block>>>(
						bufferSize,
						this->lpInputBufferCount[inputNum],
						this->outputBufferCount,
						i_lppInputBuffer[inputNum],
						i_lppOutputBuffer,
						o_lppDInputBuffer[inputNum],
						i_lppDOutputBuffer);
				}

				hipDeviceSynchronize();
			}
		}
#else
		// ���͌덷�M���z���Device�ɃR�s�[
		hipMemcpy(thrust::raw_pointer_cast(&this->lppInputBuffer_d[0]), i_lppInputBuffer, sizeof(F32*)*this->lppInputBuffer_d.size(), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(&this->lppDInputBuffer_d[0]), o_lppDInputBuffer, sizeof(F32*)*this->lppInputBuffer_d.size(), hipMemcpyHostToDevice);

		// �v�Z
		dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->bufferCountPerCh + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		device_CalculateDInput<<<grid, block>>>(
			thrust::raw_pointer_cast(&this->lppDInputBuffer_d[0]),
			thrust::raw_pointer_cast(&this->lppInputBuffer_d[0]),
			thrust::raw_pointer_cast(&this->lpInputChCount_d[0]), this->GetInputDataCount(),
			i_lppDOutputBuffer,
			i_lppOutputBuffer,
			this->bufferCountPerCh,
			loopCount);
#endif


#ifdef _DEBUG
		std::vector<std::vector<float>> lpTmpInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpInputBuffer[i][0], i_lppInputBuffer[i], sizeof(float)*lpTmpInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}

		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], i_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<std::vector<float>> lpTmpDInputBuffer(this->GetInputDataCount());
		for(int i=0; i<lpTmpInputBuffer.size(); i++)
		{
			lpTmpDInputBuffer[i].resize(this->GetBatchSize() * this->lpInputBufferCount[i]);
			hipMemcpy(&lpTmpDInputBuffer[i][0], o_lppDInputBuffer[i], sizeof(float)*lpTmpDInputBuffer[i].size(), hipMemcpyDeviceToHost);
		}
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeMultiply_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}



} // Gravisbell;
} // Layer;
} // NeuralNetwork;
