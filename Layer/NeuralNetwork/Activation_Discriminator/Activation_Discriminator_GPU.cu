//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// CPU�����p
//======================================
#include"stdafx.h"

#include"Activation_Discriminator_DATA.hpp"
#include"Activation_Discriminator_FUNC.hpp"
#include"Activation_Discriminator_Base.h"

#include"Activation_Discriminator_GPU.cuh"
#include"Activation_Discriminator_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	Activation_Discriminator_GPU::Activation_Discriminator_GPU(Gravisbell::GUID guid, Activation_Discriminator_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	Activation_Discriminator_Base	(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)		/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)		/**< �o�̓o�b�t�@�� */
		,	cudnnHandle		(NULL)
		,	inputTensorDesc	(NULL)
		,	outputTensorDesc	(NULL)
	{
		hipblasCreate(&cublasHandle);
		hipdnnCreate(&cudnnHandle);
		hipdnnCreateTensorDescriptor(&inputTensorDesc);
		hipdnnCreateTensorDescriptor(&outputTensorDesc);
		hipdnnCreateTensorDescriptor(&tmpOutputTensorDesc);
	}
	/** �f�X�g���N�^ */
	Activation_Discriminator_GPU::~Activation_Discriminator_GPU()
	{
		if(inputTensorDesc)		hipdnnDestroyTensorDescriptor(inputTensorDesc);
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(outputTensorDesc);
		if(outputTensorDesc)	hipdnnDestroyTensorDescriptor(tmpOutputTensorDesc);
		if(cudnnHandle)			hipdnnDestroy(cudnnHandle);
		if(cublasHandle)		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 Activation_Discriminator_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode Activation_Discriminator_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	ILayerData& Activation_Discriminator_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const ILayerData& Activation_Discriminator_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_Discriminator_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_Discriminator_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �ꎞ�o�̓o�b�t�@���쐬
		this->lpTmpOutputBuffer_d.resize(this->GetBatchSize() * this->inputBufferCount);
		this->lpDInputBuffer_h.resize(this->GetBatchSize() * this->inputBufferCount);
		{
			int n = this->GetBatchSize();
			int c = this->GetInputDataStruct().ch;
			int h = this->GetInputDataStruct().z * this->GetInputDataStruct().y;
			int w = this->GetInputDataStruct().x;

			const int nDims = 4;
			int dimA[nDims] = {n, c, h, w};
			int strideA[nDims] = {c*h*w, h*w, w, 1};

			hipdnnStatus_t err = hipdnnSetTensorNdDescriptor(this->outputTensorDesc,
				HIPDNN_DATA_FLOAT,
				4,
				dimA,
				strideA );

			if(err != 0)
				return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

			err = hipdnnSetTensorNdDescriptor(this->tmpOutputTensorDesc,
				HIPDNN_DATA_FLOAT,
				4,
				dimA,
				strideA );

			if(err != 0)
				return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;

			// ���̓o�b�t�@�������T�C�Y�Ȃ̂Ńf�B�X�N���v�^������Ă���
			err = hipdnnSetTensorNdDescriptor(this->inputTensorDesc,
				HIPDNN_DATA_FLOAT,
				4,
				dimA,
				strideA );

			if(err != 0)
				return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;
		}

		// �o�̓o�b�t�@���쐬
		this->lpDOutputBuffer_h.resize(this->GetBatchSize() * this->outputBufferCount);	/**< �o�͌덷�o�b�t�@��CPU���A�h���X */
		{
			int n = this->GetBatchSize();
			int c = this->GetOutputDataStruct().ch;
			int h = this->GetOutputDataStruct().z * this->GetOutputDataStruct().y;
			int w = this->GetOutputDataStruct().x;

			const int nDims = 4;
			int dimA[nDims] = {n, c, h, w};
			int strideA[nDims] = {c*h*w, h*w, w, 1};

			hipdnnStatus_t err = hipdnnSetTensorNdDescriptor(this->outputTensorDesc,
				HIPDNN_DATA_FLOAT,
				4,
				dimA,
				strideA );

			if(err != 0)
				return ErrorCode::ERROR_CODE_CUDA_INITIALIZE;
		}


		return ErrorCode::ERROR_CODE_NONE;
	}

	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Activation_Discriminator_GPU::PreProcessLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode Activation_Discriminator_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		F32 alpha = 1.0f;
		F32 beta = 0.0f;
		hipdnnStatus_t err =	hipdnnSoftmaxForward(
				this->cudnnHandle,
				HIPDNN_SOFTMAX_ACCURATE,
				HIPDNN_SOFTMAX_MODE_INSTANCE,
				&alpha,
				this->inputTensorDesc,
				i_lppInputBuffer,
				&beta,
				this->tmpOutputTensorDesc,
				thrust::raw_pointer_cast(&this->lpTmpOutputBuffer_d[0]));
		if(err != 0)
			return ErrorCode::ERROR_CODE_CUDA_CALCULATE;

		hipblasStatus_t err_cublas =	hipblasScopy(this->cublasHandle,
			this->GetBatchSize(),
			thrust::raw_pointer_cast(&this->lpTmpOutputBuffer_d[0]),
			this->inputBufferCount,
			o_lppOutputBuffer,
			1);
		if(err_cublas != 0)
			return ErrorCode::ERROR_CODE_CUDA_CALCULATE;


		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Activation_Discriminator_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			// �o�͌덷���z�X�g���ɃR�s�[
			hipMemcpy(thrust::raw_pointer_cast(&this->lpDOutputBuffer_h[0]), i_lppDOutputBuffer, sizeof(F32)*this->lpDOutputBuffer_h.size(), hipMemcpyDeviceToHost);

			// ���͌덷���v�Z
			for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
			{
				//this->lpDInputBuffer_h[batchNum*this->inputBufferCount + 0] = (       this->lpOutputBuffer_d[batchNum]) *  this->lpDOutputBuffer_h[batchNum];
				//this->lpDInputBuffer_h[batchNum*this->inputBufferCount + 1] = (1.0f - this->lpOutputBuffer_d[batchNum]) * -this->lpDOutputBuffer_h[batchNum];
				this->lpDInputBuffer_h[batchNum*this->inputBufferCount + 0] =  this->lpDOutputBuffer_h[batchNum];
				this->lpDInputBuffer_h[batchNum*this->inputBufferCount + 1] = -this->lpDOutputBuffer_h[batchNum];
			}

			hipMemcpy(o_lppDInputBuffer, thrust::raw_pointer_cast(&this->lpDInputBuffer_h[0]), sizeof(F32)*this->inputBufferCount*this->GetBatchSize(), hipMemcpyHostToDevice);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K�덷���v�Z����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Activation_Discriminator_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
