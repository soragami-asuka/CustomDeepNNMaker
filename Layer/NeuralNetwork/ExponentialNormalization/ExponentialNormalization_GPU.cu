#include "hip/hip_runtime.h"
//======================================
// �o�b�`���K�����C���[
// GPU�����p
//======================================
#include"stdafx.h"

#include<algorithm>

#include"ExponentialNormalization_DATA.hpp"
#include"ExponentialNormalization_FUNC.hpp"
#include"ExponentialNormalization_Base.h"

#include"ExponentialNormalization_GPU.cuh"
#include"ExponentialNormalization_LayerData_GPU.cuh"

#define WORKSPACE_CODE			L"WorkSpace"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {
	
#define THREAD_PER_BLOCK	32

	/** �o�͂��v�Z����
		dim = <ch,1,1>
		block = <32,1,1>
	*/
	__global__ void device_CalculateOutput(F32* o_lpOutput, const F32* i_lpInput,  const F32* i_lpAverage, const F32* i_lpVariance, U32 i_inputCountPerChannel, U32 i_loopCount, F32 i_epsilon)
	{
		U32 ch = blockIdx.x;
		U32 batchNum = blockIdx.y;
		U32 chCount = gridDim.x;

		U32 tid = threadIdx.x;

		F32 average = i_lpAverage[ch];
		F32 deviation = sqrtf(i_lpVariance[ch] + i_epsilon);

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = tid*i_loopCount + loopNum;
			if(bufferPos >= i_inputCountPerChannel)
				continue;

			U32 offset = (batchNum * chCount + ch) * i_inputCountPerChannel + bufferPos;

			F32 value = i_lpInput[offset];

			o_lpOutput[offset] = (value - average) / deviation;
		}
	}

	/** �o�͂��v�Z����
		dim = <ch,1,1>
		block = <32,1,1>
	*/
	__global__ void device_CalculateDInput(F32* o_lpDInput, const F32* i_lpDOutput, const F32* i_lpVariance, U32 i_inputCountPerChannel, U32 i_loopCount, F32 i_epsilon)
	{
		U32 ch = blockIdx.x;
		U32 batchNum = blockIdx.y;
		U32 chCount = gridDim.x;

		U32 tid = threadIdx.x;

		F32 deviation = sqrtf(i_lpVariance[ch] + i_epsilon);

		for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
		{
			U32 bufferPos = tid*i_loopCount + loopNum;
			if(bufferPos >= i_inputCountPerChannel)
				continue;

			U32 offset = (batchNum * chCount + ch) * i_inputCountPerChannel + bufferPos;

			F32 value = i_lpDOutput[offset];

			o_lpDInput[offset] = value / deviation;
		}
	}

	/** CH���Ƃ̕��ς����߂čX�V����
		dim = <ch,1,1>
		block = <32,1,1>
	*/
	__global__ void device_UpdateChAverage(F32* o_lpAverage, const F32* i_lpInputValue, U32 i_inputCountPerChannel, U32 i_batchSize, U32 i_loopCount, F32 i_alpha)
	{
		__shared__ F32 lpTmpSumValue[THREAD_PER_BLOCK];

		U32 chNum = blockIdx.x;
		U32 chCount = gridDim.x;
		U32 tid = threadIdx.x;
		U32 inputCount = chCount * i_inputCountPerChannel;

		// DWeight��Vector�̏�Z���v�Z
		lpTmpSumValue[tid] = 0.0f;
		for(U32 batchNum=0; batchNum<i_batchSize; batchNum++)
		{
			for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
			{
				U32 inputNum = THREAD_PER_BLOCK * loopNum + tid;
				if(inputNum >= i_inputCountPerChannel)
					continue;

				U32 offset = batchNum * inputCount + chNum * i_inputCountPerChannel + inputNum;

				lpTmpSumValue[tid] += i_lpInputValue[offset];
			}
		}
		__syncthreads();

		// ���v
		if(tid < 16)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 16];
		__syncthreads();
		if(tid < 8)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 8];
		__syncthreads();
		if(tid < 4)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 4];
		__syncthreads();
		if(tid < 2)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 2];
		__syncthreads();
		if(tid < 1)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 1];
		__syncthreads();

		if(tid == 0)
		{
			F32 average = lpTmpSumValue[tid] / (i_inputCountPerChannel * i_batchSize);

			o_lpAverage[chNum] = i_alpha * average + (1.0f - i_alpha) * o_lpAverage[chNum];
		}
	}

	/** CH���Ƃ̕��U�����߂čX�V����
		dim = <ch,1,1>
		block = <32,1,1>
	*/
	__global__ void device_UpdateChVariance(F32* o_lpVariance, const F32* i_lpAverage, const F32* i_lpInputValue, U32 i_inputCountPerChannel, U32 i_batchSize, U32 i_loopCount, F32 i_alpha)
	{
		__shared__ F32 lpTmpSumValue[THREAD_PER_BLOCK];

		U32 chNum = blockIdx.x;
		U32 chCount = gridDim.x;
		U32 tid = threadIdx.x;
		U32 inputCount = chCount * i_inputCountPerChannel;
		F32 average = i_lpAverage[chNum];

		// DWeight��Vector�̏�Z���v�Z
		lpTmpSumValue[tid] = 0.0f;
		for(U32 batchNum=0; batchNum<i_batchSize; batchNum++)
		{
			for(U32 loopNum=0; loopNum<i_loopCount; loopNum++)
			{
				U32 inputNum = THREAD_PER_BLOCK * loopNum + tid;
				if(inputNum >= i_inputCountPerChannel)
					continue;

				U32 offset = batchNum * inputCount + chNum * i_inputCountPerChannel + inputNum;

				lpTmpSumValue[tid] += (i_lpInputValue[offset] - average) * (i_lpInputValue[offset] - average);
			}
		}
		__syncthreads();

		// ���v
		if(tid < 16)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 16];
		__syncthreads();
		if(tid < 8)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 8];
		__syncthreads();
		if(tid < 4)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 4];
		__syncthreads();
		if(tid < 2)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 2];
		__syncthreads();
		if(tid < 1)
			lpTmpSumValue[tid] += lpTmpSumValue[tid + 1];
		__syncthreads();

		if(tid == 0)
		{
			F32 variance = lpTmpSumValue[tid] / (i_inputCountPerChannel * i_batchSize);

			o_lpVariance[chNum] = i_alpha * variance + (1.0f - i_alpha) * o_lpVariance[chNum];
		}
	}

	/** �R���X�g���N�^ */
	ExponentialNormalization_GPU::ExponentialNormalization_GPU(Gravisbell::GUID guid, ExponentialNormalization_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	ExponentialNormalization_Base	(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData				(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount		(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount		(0)				/**< �o�̓o�b�t�@�� */
		,	channeclBufferCount		(0)				/**< 1�`�����l��������̃o�b�t�@�� */
		,	temporaryMemoryManager	(i_temporaryMemoryManager)
	{
	}
	/** �f�X�g���N�^ */
	ExponentialNormalization_GPU::~ExponentialNormalization_GPU()
	{
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 ExponentialNormalization_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode ExponentialNormalization_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	ExponentialNormalization_LayerData_Base& ExponentialNormalization_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const ExponentialNormalization_LayerData_Base& ExponentialNormalization_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode ExponentialNormalization_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode ExponentialNormalization_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �`�����l�����Ƃ̃o�b�t�@�����m�F
		this->channeclBufferCount = this->GetInputDataStruct().z * this->GetInputDataStruct().y * this->GetInputDataStruct().x;
		if(this->channeclBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;;

		return ErrorCode::ERROR_CODE_NONE;
	}

	
	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode ExponentialNormalization_GPU::PreProcessLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode ExponentialNormalization_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		// �v�Z
		dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->channeclBufferCount + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		device_CalculateOutput<<<grid,block>>>(
			o_lppOutputBuffer,
			i_lppInputBuffer,
			thrust::raw_pointer_cast(&this->layerData.lpMean[0]),
			thrust::raw_pointer_cast(&this->layerData.lpVariance[0]),
			this->channeclBufferCount,
			loopCount,
			this->layerData.layerStructure.epsilon);
		
#ifdef _DEBUG
			std::vector<float> lpTmpInputBuffer(this->GetBatchSize() * this->inputBufferCount);
			hipMemcpy(&lpTmpInputBuffer[0], i_lppInputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

			std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
			hipMemcpy(&lpTmpOutputBuffer[0], o_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode ExponentialNormalization_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		if(o_lppDInputBuffer)
		{
			// �v�Z
			dim3 grid(this->GetOutputDataStruct().ch, this->GetBatchSize());
			dim3 block(THREAD_PER_BLOCK);
			U32 loopCount = (this->channeclBufferCount + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

			device_CalculateDInput<<<grid,block>>>(
				o_lppDInputBuffer,
				i_lppDOutputBuffer,
				thrust::raw_pointer_cast(&this->layerData.lpVariance[0]),
				this->channeclBufferCount,
				loopCount,
				this->layerData.layerStructure.epsilon);


#ifdef _DEBUG
			std::vector<float> lpTmpInputBuffer(this->GetBatchSize() * this->inputBufferCount);
			hipMemcpy(&lpTmpInputBuffer[0], i_lppInputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

			std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
			hipMemcpy(&lpTmpOutputBuffer[0], i_lppOutputBuffer, sizeof(float)*lpTmpOutputBuffer.size(), hipMemcpyDeviceToHost);

			std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
			hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

			std::vector<float> lpTmpDInputBuffer(this->GetBatchSize() * this->inputBufferCount);
			hipMemcpy(&lpTmpDInputBuffer[0], o_lppDInputBuffer, sizeof(float)*lpTmpDInputBuffer.size(), hipMemcpyDeviceToHost);
#endif
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode ExponentialNormalization_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(o_lppDInputBuffer)
		{
			CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
		}

		// �w�K�񐔂��X�V
		this->layerData.learnTime++;

		// CH���Ƃɕ��ςƕ��U�����߂�
		F32 alpha = 0.0f;
		if(this->layerData.learnTime < this->layerData.layerStructure.InitParameterTime)
			alpha = 1.0f / (this->layerData.learnTime + 1);
		else
			alpha = std::min<F32>(1.0f, this->GetRuntimeParameterByStructure().AccelCoeff * 2 / (this->layerData.layerStructure.ExponentialTime + 1));

		dim3 grid(this->GetOutputDataStruct().ch);
		dim3 block(THREAD_PER_BLOCK);
		U32 loopCount = (this->channeclBufferCount + (THREAD_PER_BLOCK-1)) / THREAD_PER_BLOCK;

		// ���ς��X�V����
		device_UpdateChAverage<<<grid,block>>>(
			thrust::raw_pointer_cast(&this->layerData.lpMean[0]),
			i_lppInputBuffer,
			this->channeclBufferCount,
			this->GetBatchSize(),
			loopCount,
			alpha);

		// ���U���X�V����
		device_UpdateChVariance<<<grid,block>>>(
			thrust::raw_pointer_cast(&this->layerData.lpVariance[0]),
			thrust::raw_pointer_cast(&this->layerData.lpMean[0]),
			i_lppInputBuffer,
			this->channeclBufferCount,
			this->GetBatchSize(),
			loopCount,
			alpha);


#ifdef _DEBUG
		std::vector<float> lpTmpInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpInputBuffer[0], i_lppInputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpOutputBuffer[0], i_lppOutputBuffer, sizeof(float)*lpTmpInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDOutputBuffer(this->GetBatchSize() * this->outputBufferCount);
		hipMemcpy(&lpTmpDOutputBuffer[0], i_lppDOutputBuffer, sizeof(float)*lpTmpDOutputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpTmpDInputBuffer(this->GetBatchSize() * this->inputBufferCount);
		hipMemcpy(&lpTmpDInputBuffer[0], o_lppDInputBuffer, sizeof(float)*lpTmpDInputBuffer.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpMean(this->layerData.lpMean.size());
		hipMemcpy(&lpMean[0], thrust::raw_pointer_cast(&this->layerData.lpMean[0]), sizeof(float)*lpMean.size(), hipMemcpyDeviceToHost);

		std::vector<float> lpVariance(this->layerData.lpVariance.size());
		hipMemcpy(&lpVariance[0], thrust::raw_pointer_cast(&this->layerData.lpVariance[0]), sizeof(float)*lpVariance.size(), hipMemcpyDeviceToHost);
#endif

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
