//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̏������C���[
// �����̃��C���[�����A��������
// GPU����
//======================================
#include"stdafx.h"

#include"FeedforwardNeuralNetwork_Base.h"
#include"FeedforwardNeuralNetwork_GPU.cuh"

// CUDA�p
#pragma warning(push)
#pragma warning(disable : 4267)
#include <hip/hip_runtime_api.h>
#pragma warning(pop)


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

	/** �R���X�g���N�^ */
	FeedforwardNeuralNetwork_GPU::FeedforwardNeuralNetwork_GPU(const Gravisbell::GUID& i_guid, class FeedforwardNeuralNetwork_LayerData_Base& i_layerData)
		:	FeedforwardNeuralNetwork_Base(i_guid, i_layerData)
	{
	}
	/** �f�X�g���N�^ */
	FeedforwardNeuralNetwork_GPU::~FeedforwardNeuralNetwork_GPU()
	{
	}

	/** ���C���[��ʂ̎擾.
		ELayerKind �̑g�ݍ��킹. */
	U32 FeedforwardNeuralNetwork_GPU::GetLayerKind(void)const
	{
		return this->GetLayerKindBase() | Gravisbell::Layer::LAYER_KIND_GPU;
	}


	//====================================
	// ���o�̓o�b�t�@�֘A
	//====================================
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode FeedforwardNeuralNetwork_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, FeedforwardNeuralNetwork_Base::GetOutputBuffer(), sizeof(F32)*outputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode FeedforwardNeuralNetwork_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, FeedforwardNeuralNetwork_Base::GetDInputBuffer(), sizeof(F32)*inputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


}	// NeuralNetwork
}	// Layer
}	// Gravisbell

