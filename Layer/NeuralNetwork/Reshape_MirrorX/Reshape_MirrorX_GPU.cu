#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"Reshape_MirrorX_DATA.hpp"
#include"Reshape_MirrorX_FUNC.hpp"
#include"Reshape_MirrorX_Base.h"

#include"Reshape_MirrorX_GPU.cuh"
#include"Reshape_MirrorX_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	Reshape_MirrorX_GPU::Reshape_MirrorX_GPU(Gravisbell::GUID guid, Reshape_MirrorX_LayerData_GPU& i_layerData, const IODataStruct& i_inputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	Reshape_MirrorX_Base				(guid, i_inputDataStruct, i_layerData.GetOutputDataStruct(&i_inputDataStruct, 1))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
	{
	}
	/** �f�X�g���N�^ */
	Reshape_MirrorX_GPU::~Reshape_MirrorX_GPU()
	{
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 Reshape_MirrorX_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode Reshape_MirrorX_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	Reshape_MirrorX_LayerData_Base& Reshape_MirrorX_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const Reshape_MirrorX_LayerData_Base& Reshape_MirrorX_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_MirrorX_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// �o�͌덷�o�b�t�@
		this->m_lpDOutputBuffer_h.resize(this->GetBatchSize() * this->outputBufferCount);
		this->m_lppDOutputBuffer.resize(this->GetBatchSize());
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
			this->m_lppDOutputBuffer[batchNum] = &this->m_lpDOutputBuffer_h[batchNum * this->outputBufferCount];

		// ���͌덷�o�b�t�@
		this->m_lpDInputBuffer_h.resize(this->GetBatchSize() * this->inputBufferCount);
		this->m_lppDInputBuffer.resize(this->GetBatchSize());
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
			this->m_lppDInputBuffer[batchNum] = &this->m_lpDInputBuffer_h[batchNum * this->inputBufferCount];


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_MirrorX_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// ���̓o�b�t�@�ۑ��p�̃A�h���X�z����쐬
		this->m_lpInputBuffer_h.resize(this->inputBufferCount * this->GetBatchSize());
		this->m_lppInputBuffer.resize(this->GetBatchSize(), NULL);
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
		{
			this->m_lppInputBuffer[batchNum] = &this->m_lpInputBuffer_h[batchNum * this->inputBufferCount];
		}

		// �o�̓o�b�t�@���쐬
		this->m_lpOutputBuffer_h.resize(this->GetBatchSize() * this->outputBufferCount);
		this->m_lppOutputBuffer.resize(this->GetBatchSize());
		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
		{
			this->m_lppOutputBuffer[batchNum] = &this->m_lpOutputBuffer_h[batchNum * this->outputBufferCount];
		}

		return ErrorCode::ERROR_CODE_NONE;
	}



	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode Reshape_MirrorX_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode Reshape_MirrorX_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		hipMemcpy(&this->m_lpInputBuffer_h[0], i_lppInputBuffer, sizeof(F32)*this->inputBufferCount*this->GetBatchSize(), hipMemcpyDeviceToHost);

		for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
		{
			for(U32 ch=0; ch<this->GetInputDataStruct().ch; ch++)
			{
				for(U32 inputZ=0; inputZ<this->GetInputDataStruct().z; inputZ++)
				{
					for(U32 inputY=0; inputY<this->GetInputDataStruct().y; inputY++)
					{
						for(U32 inputX=0; inputX<this->GetInputDataStruct().x; inputX++)
						{
							U32 inputOffset   = this->GetInputDataStruct().POSITION_TO_OFFSET(inputX, inputY, inputZ, ch);

							U32 outputOffset0 = this->GetOutputDataStruct().POSITION_TO_OFFSET(this->GetInputDataStruct().x-1-inputX, inputY, inputZ, ch);
							U32 outputOffset1 = this->GetOutputDataStruct().POSITION_TO_OFFSET(this->GetInputDataStruct().x-1+inputX, inputY, inputZ, ch);

							this->m_lppOutputBuffer[batchNum][outputOffset0] = this->m_lppInputBuffer[batchNum][inputOffset];
							this->m_lppOutputBuffer[batchNum][outputOffset1] = this->m_lppInputBuffer[batchNum][inputOffset];
						}
					}
				}
			}
		}

		// �o�̓o�b�t�@���f�o�C�X�ɃR�s�[
		hipMemcpy(o_lppOutputBuffer, &this->m_lpOutputBuffer_h[0], sizeof(F32)*this->outputBufferCount*this->GetBatchSize(), hipMemcpyHostToDevice);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v�Ȕz���[GetOutputDataCount()]�z��
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Reshape_MirrorX_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		// ���͌덷�v�Z
		if(i_lppDOutputBuffer && o_lppDInputBuffer)
		{
			// �o�͌덷�o�b�t�@���z�X�g�ɃR�s�[
			hipMemcpy(&this->m_lpDOutputBuffer_h[0], i_lppDOutputBuffer, sizeof(F32)*this->outputBufferCount*this->GetBatchSize(), hipMemcpyDeviceToHost);

			// ���͌덷��������
			memset(&this->m_lpDInputBuffer_h[0], 0, sizeof(F32)*this->GetBatchSize()*this->inputBufferCount);

			// ���͌덷�v�Z
			for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
			{
				for(U32 ch=0; ch<this->GetOutputDataStruct().ch; ch++)
				{
					for(U32 outputZ=0; outputZ<this->GetOutputDataStruct().z; outputZ++)
					{
						for(U32 outputY=0; outputY<this->GetOutputDataStruct().y; outputY++)
						{
							for(U32 outputX=0; outputX<this->GetOutputDataStruct().x; outputX++)
							{
								U32 inputX = abs((S32)(outputX - (this->GetInputDataStruct().x-1)));

								U32 outputOffset  = this->GetOutputDataStruct().POSITION_TO_OFFSET(outputX, outputY, outputZ, ch);
								U32 inputOffset   = this->GetInputDataStruct().POSITION_TO_OFFSET(inputX,  outputY, outputZ, ch);

								this->m_lppDInputBuffer[batchNum][inputOffset] += this->m_lppDOutputBuffer[batchNum][outputOffset];
							}
						}
					}
				}
			}

			// ���͌덷���f�o�C�X�ɃR�s�[
			hipMemcpy(o_lppDInputBuffer, &this->m_lpDInputBuffer_h[0], sizeof(F32)*this->inputBufferCount*this->GetBatchSize(), hipMemcpyHostToDevice);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode Reshape_MirrorX_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer, BATCH_BUFFER_POINTER o_lppDInputBuffer, CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
