#include "hip/hip_runtime.h"
//======================================
// �S�����j���[�����l�b�g���[�N�̃��C���[�f�[�^
// GPU����
//======================================
#include"stdafx.h"

#include"FullyConnect_LayerData_GPU.cuh"
#include"FullyConnect_FUNC.hpp"
#include"FullyConnect_GPU.cuh"

#include"RandomUtility.h"

#pragma warning(push)
#pragma warning(disable : 4267)
#include <hip/hip_runtime.h> // need CUDA_VERSION
#include <hipDNN.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include ""
#pragma warning(pop)

#include"Library/NeuralNetwork/Optimizer.h"


namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	//===========================
	// �R���X�g���N�^ / �f�X�g���N�^
	//===========================
	/** �R���X�g���N�^ */
	FullyConnect_LayerData_GPU::FullyConnect_LayerData_GPU(const Gravisbell::GUID& guid)
		:	FullyConnect_LayerData_Base(guid)
	{
	}
	/** �f�X�g���N�^ */
	FullyConnect_LayerData_GPU::~FullyConnect_LayerData_GPU()
	{
	}


	//===========================
	// ������
	//===========================
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode FullyConnect_LayerData_GPU::Initialize(void)
	{
		// �����Œ艻
		Utility::Random::Initialize(0);

		// ���̓o�b�t�@�����m�F
		unsigned int inputBufferCount = this->GetInputBufferCount();
		if(inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_COMMON_OUT_OF_VALUERANGE;

		// �j���[���������m�F
		unsigned int neuronCount = this->GetNeuronCount();
		if(neuronCount == 0)
			return ErrorCode::ERROR_CODE_COMMON_OUT_OF_VALUERANGE;

		// �o�b�t�@���m�ۂ��A�����l��ݒ�
		this->lppNeuron_d.reserve(neuronCount * inputBufferCount);
		this->lppNeuron_d.resize(neuronCount * inputBufferCount);
		this->lpBias_d.reserve(neuronCount);
		this->lpBias_d.resize(neuronCount);

		thrust::host_vector<F32> lpTmpNeuron(neuronCount * inputBufferCount);
		thrust::host_vector<F32> lpTmpBias(neuronCount);

		float maxArea = sqrt(6.0f / (inputBufferCount + neuronCount));
		for(U32 i=0; i<lpTmpNeuron.size(); i++)
			lpTmpNeuron[i] = ((F32)Utility::Random::GetValue() - 0.5f) * 2.0f * maxArea;
		for(U32 i=0; i<lpTmpBias.size(); i++)
			lpTmpBias[i] = ((F32)Utility::Random::GetValue() - 0.5f) * 2.0f * maxArea;

		this->lppNeuron_d = lpTmpNeuron;
		this->lpBias_d = lpTmpBias;

		return ErrorCode::ERROR_CODE_NONE;
	}
	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@param	i_config			�ݒ���
		@oaram	i_inputDataStruct	���̓f�[�^�\�����
		@return	���������ꍇ0 */
	ErrorCode FullyConnect_LayerData_GPU::Initialize(const SettingData::Standard::IData& i_data, const IODataStruct& i_inputDataStruct)
	{
		ErrorCode err;

		// �ݒ���̓o�^
		err = this->SetLayerConfig(i_data);
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		// ���̓f�[�^�\���̐ݒ�
		this->inputDataStruct = i_inputDataStruct;

		// ������
		err = this->Initialize();
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		// �I�v�e�B�}�C�U�[�̐ݒ�
		err = this->ChangeOptimizer(L"SGD");
		if(err != ErrorCode::ERROR_CODE_NONE)
			return err;

		return this->Initialize();
	}
	/** ������. �o�b�t�@����f�[�^��ǂݍ���
		@param i_lpBuffer	�ǂݍ��݃o�b�t�@�̐擪�A�h���X.
		@param i_bufferSize	�ǂݍ��݉\�o�b�t�@�̃T�C�Y.
		@return	���������ꍇ0 */
	ErrorCode FullyConnect_LayerData_GPU::InitializeFromBuffer(const BYTE* i_lpBuffer, U32 i_bufferSize, S32& o_useBufferSize )
	{
		int readBufferByte = 0;

		// ���̓f�[�^�\��
		memcpy(&this->inputDataStruct, &i_lpBuffer[readBufferByte], sizeof(this->inputDataStruct));
		readBufferByte += sizeof(this->inputDataStruct);

		// �ݒ���
		S32 useBufferByte = 0;
		SettingData::Standard::IData* pLayerStructure = CreateLayerStructureSettingFromBuffer(&i_lpBuffer[readBufferByte], i_bufferSize, useBufferByte);
		if(pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_INITLAYER_READ_CONFIG;
		readBufferByte += useBufferByte;
		this->SetLayerConfig(*pLayerStructure);
		delete pLayerStructure;

		// ����������
		this->Initialize();

		// �o�b�t�@����R�s�[
		// �j���[����
		hipMemcpy(
			thrust::raw_pointer_cast(&this->lppNeuron_d[0]),
			&i_lpBuffer[readBufferByte],
			sizeof(F32) * this->lppNeuron_d.size(),
			hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32) * (S32)this->lppNeuron_d.size();

		// �o�C�A�X
		hipMemcpy(
			thrust::raw_pointer_cast(&this->lpBias_d[0]),
			&i_lpBuffer[readBufferByte],
			sizeof(F32) * this->lpBias_d.size(),
			hipMemcpyHostToDevice);
		readBufferByte += sizeof(F32) * (S32)this->lpBias_d.size();


		// �I�v�e�B�}�C�U
		S32 useBufferSize = 0;
		// bias
		if(this->m_pOptimizer_bias)
			delete this->m_pOptimizer_bias;
		this->m_pOptimizer_bias = CreateOptimizerFromBuffer_GPU(&i_lpBuffer[readBufferByte], i_bufferSize-readBufferByte, useBufferSize);
		readBufferByte += useBufferSize;
		// neuron
		if(this->m_pOptimizer_neuron)
			delete this->m_pOptimizer_neuron;
		this->m_pOptimizer_neuron = CreateOptimizerFromBuffer_GPU(&i_lpBuffer[readBufferByte], i_bufferSize-readBufferByte, useBufferSize);
		readBufferByte += useBufferSize;


		o_useBufferSize = readBufferByte;

		return ErrorCode::ERROR_CODE_NONE;
	}


	//===========================
	// ���C���[�ۑ�
	//===========================
	/** ���C���[���o�b�t�@�ɏ�������.
		@param o_lpBuffer	�������ݐ�o�b�t�@�̐擪�A�h���X. GetUseBufferByteCount�̖߂�l�̃o�C�g�����K�v
		@return ���������ꍇ�������񂾃o�b�t�@�T�C�Y.���s�����ꍇ�͕��̒l */
	S32 FullyConnect_LayerData_GPU::WriteToBuffer(BYTE* o_lpBuffer)const
	{
		if(this->pLayerStructure == NULL)
			return ErrorCode::ERROR_CODE_NONREGIST_CONFIG;

		int writeBufferByte = 0;

		// ���̓f�[�^�\��
		memcpy(&o_lpBuffer[writeBufferByte], &this->inputDataStruct, sizeof(this->inputDataStruct));
		writeBufferByte += sizeof(this->inputDataStruct);

		// �ݒ���
		writeBufferByte += this->pLayerStructure->WriteToBuffer(&o_lpBuffer[writeBufferByte]);

		// �j���[����
		hipMemcpy(
			&o_lpBuffer[writeBufferByte],
			thrust::raw_pointer_cast(&this->lppNeuron_d[0]),
			sizeof(F32) * this->lppNeuron_d.size(),
			hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32) * (S32)this->lppNeuron_d.size();

		// �o�C�A�X
		hipMemcpy(
			&o_lpBuffer[writeBufferByte],
			thrust::raw_pointer_cast(&this->lpBias_d[0]),
			sizeof(F32) * this->lpBias_d.size(),
			hipMemcpyDeviceToHost);
		writeBufferByte += sizeof(F32) * (S32)this->lpBias_d.size();


		// �I�v�e�B�}�C�U
		// bias
		writeBufferByte += this->m_pOptimizer_bias->WriteToBuffer(&o_lpBuffer[writeBufferByte]);
		// neuron
		writeBufferByte += this->m_pOptimizer_neuron->WriteToBuffer(&o_lpBuffer[writeBufferByte]);


		return writeBufferByte;
	}


	//===========================
	// ���C���[�쐬
	//===========================
	/** ���C���[���쐬����.
		@param guid	�V�K�������郌�C���[��GUID. */
	ILayerBase* FullyConnect_LayerData_GPU::CreateLayer(const Gravisbell::GUID& guid)
	{
		return new FullyConnect_GPU(guid, *this);
	}


	//===========================
	// �I�v�e�B�}�C�U�[�ݒ�
	//===========================
	/** �I�v�e�B�}�C�U�[��ύX���� */
	ErrorCode FullyConnect_LayerData_GPU::ChangeOptimizer(const wchar_t i_optimizerID[])
	{
		ChangeOptimizer_GPU(&this->m_pOptimizer_bias,   i_optimizerID, (U32)this->lpBias_d.size());
		ChangeOptimizer_GPU(&this->m_pOptimizer_neuron, i_optimizerID, (U32)this->lppNeuron_d.size());

		return ErrorCode::ERROR_CODE_NONE;
	}

} // Gravisbell;
} // Layer;
} // NeuralNetwork;


using namespace Gravisbell;

/** Create a layer for GPU processing.
  * @param GUID of layer to create.
  */
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPU(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const Gravisbell::SettingData::Standard::IData& i_data, const Gravisbell::IODataStruct& i_inputDataStruct)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::FullyConnect_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::FullyConnect_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	Gravisbell::ErrorCode errCode = pLayerData->Initialize(i_data, i_inputDataStruct);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	return pLayerData;
}
EXPORT_API Gravisbell::Layer::ILayerData* CreateLayerDataGPUfromBuffer(const Gravisbell::Layer::NeuralNetwork::ILayerDLLManager* pLayerDLLManager, Gravisbell::GUID guid, const BYTE* i_lpBuffer, S32 i_bufferSize, S32& o_useBufferSize)
{
	// �쐬
	Gravisbell::Layer::NeuralNetwork::FullyConnect_LayerData_GPU* pLayerData = new Gravisbell::Layer::NeuralNetwork::FullyConnect_LayerData_GPU(guid);
	if(pLayerData == NULL)
		return NULL;

	// ������
	S32 useBufferSize = 0;
	Gravisbell::ErrorCode errCode = pLayerData->InitializeFromBuffer(i_lpBuffer, i_bufferSize, useBufferSize);
	if(errCode != Gravisbell::ErrorCode::ERROR_CODE_NONE)
	{
		delete pLayerData;
		return NULL;
	}

	// �g�p�����o�b�t�@�ʂ��i�[
	o_useBufferSize = useBufferSize;

	return pLayerData;
}