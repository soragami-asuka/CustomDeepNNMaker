#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// GPU�����p
//======================================
#include"stdafx.h"

#include"FullyConnect_DATA.hpp"
#include"FullyConnect_FUNC.hpp"
#include"FullyConnect_Base.h"

#include"FullyConnect_GPU.cuh"
#include"FullyConnect_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define BLOCK_SIZE	(16)

namespace
{
	/** �x�N�g���̗v�f���m�̊|���Z. */
	__global__ void cuda_func_multiplVector(const F32* i_lpInputBufferA, const F32* i_lpInputBufferB, F32* o_lpOutputBuffer, U32 i_bufferSize)
	{
		const U32 bufferPos = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		if(bufferPos >= i_bufferSize)	// ���򂷂邪������warp�����Ȃ̂ŁA�������x�ɉe���͂Ȃ��͂�...
			return;

		o_lpOutputBuffer[bufferPos] = i_lpInputBufferA[bufferPos] * i_lpInputBufferB[bufferPos];
	}
	/** �x�N�g���̗v�f���m�̊|���Z. */
	__global__ void cuda_func_multiplVectorWithScaler(const F32* i_lpInputBufferA, const F32* i_lpInputBufferB, F32* o_lpOutputBuffer, U32 i_bufferSize, F32 alpha, F32 beta)
	{
		const U32 bufferPos = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		if(bufferPos >= i_bufferSize)	// ���򂷂邪������warp�����Ȃ̂ŁA�������x�ɉe���͂Ȃ��͂�...
			return;

		o_lpOutputBuffer[bufferPos] = alpha * i_lpInputBufferA[bufferPos] * i_lpInputBufferB[bufferPos] + beta * o_lpOutputBuffer[bufferPos];
	}
}

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	FullyConnect_GPU::FullyConnect_GPU(Gravisbell::GUID guid, FullyConnect_LayerData_GPU& i_layerData)
		:	FullyConnect_Base	(guid)
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)		/**< ���̓o�b�t�@�� */
		,	neuronCount						(0)		/**< �j���[������ */
		,	outputBufferCount				(0)		/**< �o�̓o�b�t�@�� */
	{
		hipblasCreate(&cublasHandle);
	}
	/** �f�X�g���N�^ */
	FullyConnect_GPU::~FullyConnect_GPU()
	{
		hipblasDestroy(cublasHandle);
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 FullyConnect_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode FullyConnect_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	FullyConnect_LayerData_Base& FullyConnect_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const FullyConnect_LayerData_Base& FullyConnect_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//===========================
	// ���C���[�ۑ�
	//===========================
	/** ���C���[���o�b�t�@�ɏ�������.
		@param o_lpBuffer	�������ݐ�o�b�t�@�̐擪�A�h���X. GetUseBufferByteCount�̖߂�l�̃o�C�g�����K�v
		@return ���������ꍇ�������񂾃o�b�t�@�T�C�Y.���s�����ꍇ�͕��̒l */
	S32 FullyConnect_GPU::WriteToBuffer(BYTE* o_lpBuffer)const
	{
		return this->layerData.WriteToBuffer(o_lpBuffer);
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode FullyConnect_GPU::PreProcessLearn(U32 batchSize)
	{
		ErrorCode errorCode = this->PreProcessCalculate(batchSize);
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// ���͍����o�b�t�@���쐬
		this->lpDInputBuffer_d.resize(this->batchSize * this->inputBufferCount);

		// �o�C�A�X�X�V�p�̃x�N�g�����쐬
		lpBiasUpdateVector_d.resize(this->batchSize);
		{
			thrust::host_vector<F32> lpBuf(this->batchSize, 1.0f);
			this->lpBiasUpdateVector_d = lpBuf;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode FullyConnect_GPU::PreProcessCalculate(U32 batchSize)
	{
		this->batchSize = batchSize;

		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;
		
		// �j���[���������m�F
		this->neuronCount = this->GetNeuronCount();
		if(this->neuronCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_NEURON_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �j���[�����o�b�t�@�̃T�C�Y�m�F
		if(this->layerData.lppNeuron_d.size() != this->neuronCount * this->inputBufferCount)
			return ErrorCode::ERROR_CODE_FRAUD_NEURON_COUNT;

		// �o�̓o�b�t�@���쐬
		this->lpOutputBuffer_d.resize(this->batchSize * this->outputBufferCount);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K���[�v�̏���������.�f�[�^�Z�b�g�̊w�K�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode FullyConnect_GPU::PreProcessLearnLoop(const SettingData::Standard::IData& data)
	{
		if(this->pLearnData != NULL)
			delete this->pLearnData;
		this->pLearnData = data.Clone();

		// �w�K�W��
		{
			auto pItem = dynamic_cast<const Gravisbell::SettingData::Standard::IItem_Float*>(data.GetItemByID(L"LearnCoeff"));
			if(pItem)
				this->learnData.LearnCoeff = pItem->GetValue();
			else
				this->learnData.LearnCoeff = 1.0f;
		}

		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}
	/** ���Z���[�v�̏���������.�f�[�^�Z�b�g�̉��Z�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode FullyConnect_GPU::PreProcessCalculateLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode FullyConnect_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		// ���̓o�b�t�@��ۊ�
		this->m_lppInputBuffer_d = i_lpInputBuffer;

		// �o�C�A�X���o�͐M���ɃR�s�[����
		{
			for(U32 batchNum=0; batchNum<this->batchSize; batchNum++)
			{
				hipError_t err = hipMemcpy(
					thrust::raw_pointer_cast(&lpOutputBuffer_d[batchNum * this->outputBufferCount]),
					thrust::raw_pointer_cast(&this->layerData.lpBias_d[0]),
					sizeof(F32) * this->neuronCount,
					hipMemcpyDeviceToDevice);
				if(err != 0)
					return ERROR_CODE_CUDA_COPY_MEMORY;
			}
		}

		// �j���[����T�~���͐M��
		{
			// C = aAB + bC;

			F32 alpha = 1.0f;
			F32 beta  = 1.0f;	// �o�C�A�X��C�ɃR�s�[�ς݂Ȃ̂ł��̂܂ܗ��p���邽�߂�1.0���w��

			hipblasSgemm(
				this->cublasHandle,
				HIPBLAS_OP_T,
				HIPBLAS_OP_N,
				this->neuronCount,	// �s��A�̍s��
				this->batchSize,	// �s��B�̗�
				this->inputBufferCount,	// �s��A�̗�,�s��B�̍s��
				&alpha,
				thrust::raw_pointer_cast(&this->layerData.lppNeuron_d[0]),	// �s��A
				this->inputBufferCount,										// �s��A�̓]�u�O�̍s��
				i_lpInputBuffer,											// �s��B
				this->inputBufferCount,										// �s��B�̓]�u�O�̍s��
				&beta,
				thrust::raw_pointer_cast(&lpOutputBuffer_d[0]),
				this->outputBufferCount);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER FullyConnect_GPU::GetOutputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpOutputBuffer_d[0]);
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode FullyConnect_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32) * outputBufferCount * this->batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** �w�K�덷���v�Z����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode FullyConnect_GPU::CalculateLearnError(CONST_BATCH_BUFFER_POINTER i_lpDOutputBufferPrev)
	{
		// �o�͌덷�o�b�t�@�̃A�h���X��z��Ɋi�[
		this->m_lppDOutputBuffer_d = i_lpDOutputBufferPrev;

		// ���͌덷�������v�Z
		{
			F32 alpha = 1.0f;
			F32 beta  = 0.0f;

			hipblasSgemm(
				this->cublasHandle,
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				this->inputBufferCount,	// �s��A�̍s��
				this->batchSize,		// �s��B�̗�
				this->neuronCount,		// �s��A�̗�,�s��B�̍s��
				&alpha,
				thrust::raw_pointer_cast(&this->layerData.lppNeuron_d[0]),	// �s��A
				this->inputBufferCount,										// �s��A�̓]�u�O�̍s��
				this->m_lppDOutputBuffer_d,									// �s��B
				this->neuronCount,											// �s��B�̓]�u�O�̍s��
				&beta,
				thrust::raw_pointer_cast(&this->lpDInputBuffer_d[0]),
				this->inputBufferCount);
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K���������C���[�ɔ��f������.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		�o�͌덷�����A���͌덷�����͒��O��CalculateLearnError�̒l���Q�Ƃ���. */
	ErrorCode FullyConnect_GPU::ReflectionLearnError(void)
	{
		// �o�C�A�X�X�V
		{
			F32 alpha = this->learnData.LearnCoeff;
			F32 beta  = 1.0f;

			hipblasSgemm(
				this->cublasHandle,
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				this->neuronCount,		// �s��A�̍s��
				1,						// �s��B�̗�
				this->batchSize,		// �s��A�̗�,�s��B�̍s��
				&alpha,
				this->m_lppDOutputBuffer_d,	// �s��A
				this->neuronCount,											// �s��A�̓]�u�O�̍s��
				thrust::raw_pointer_cast(&this->lpBiasUpdateVector_d[0]),	// �s��B
				this->batchSize,											// �s��B�̓]�u�O�̍s��
				&beta,
				thrust::raw_pointer_cast(&this->layerData.lpBias_d[0]),
				this->neuronCount);
		}

		// �j���[�����X�V
		{
			// �j���[�����̌덷���v�Z���ĉ��Z����
			{
				F32 alpha = this->learnData.LearnCoeff;
				F32 beta  = 1.0f;

				hipblasSgemm(
					this->cublasHandle,
					HIPBLAS_OP_N,
					HIPBLAS_OP_T,
					this->inputBufferCount,	// �s��A�̍s��
					this->neuronCount,		// �s��B�̗�
					this->batchSize,		// �s��A�̗�,�s��B�̍s��
					&alpha,
					this->m_lppInputBuffer_d,		// �s��A
					this->inputBufferCount,										// �s��A�̓]�u�O�̍s��
					this->m_lppDOutputBuffer_d,	// �s��B
					this->neuronCount,										// �s��B�̓]�u�O�̍s��
					&beta,
					thrust::raw_pointer_cast(&this->layerData.lppNeuron_d[0]),
					this->inputBufferCount);
			}
		}

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER FullyConnect_GPU::GetDInputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpDInputBuffer_d[0]);
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode FullyConnect_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32) * inputBufferCount * batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
