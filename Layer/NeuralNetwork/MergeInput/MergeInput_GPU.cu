//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"MergeInput_DATA.hpp"
#include"MergeInput_FUNC.hpp"
#include"MergeInput_Base.h"

#include"MergeInput_GPU.cuh"
#include"MergeInput_LayerData_GPU.cuh"

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {


	/** �R���X�g���N�^ */
	MergeInput_GPU::MergeInput_GPU(Gravisbell::GUID guid, MergeInput_LayerData_GPU& i_layerData, const std::vector<IODataStruct>& i_lpInputDataStruct, Gravisbell::Common::ITemporaryMemoryManager& i_temporaryMemoryManager)
		:	MergeInput_Base					(guid, i_lpInputDataStruct, i_layerData.GetOutputDataStruct(&i_lpInputDataStruct[0], (U32)i_lpInputDataStruct.size()))
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
	{
	}
	/** �f�X�g���N�^ */
	MergeInput_GPU::~MergeInput_GPU()
	{
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 MergeInput_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode MergeInput_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	MergeInput_LayerData_Base& MergeInput_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const MergeInput_LayerData_Base& MergeInput_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeInput_GPU::PreProcessLearn()
	{
		ErrorCode errorCode = this->PreProcessCalculate();
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// �o�͌덷�o�b�t�@�̔z����쐬
		this->m_lppDInputBuffer.resize(this->GetInputDataCount());

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeInput_GPU::PreProcessCalculate()
	{
		// ���̓o�b�t�@�����m�F
		this->lpInputBufferCount.resize(this->GetInputDataCount());
		for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
		{
			this->lpInputBufferCount[inputNum] = this->GetInputBufferCount(inputNum);
			if(this->lpInputBufferCount[inputNum] == 0)
				return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;
		}

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���[�v�̏���������.�f�[�^�Z�b�g�̎��s�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MergeInput_GPU::PreProcessLoop()
	{
		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode MergeInput_GPU::Calculate_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppOutputBuffer)
	{
		switch(this->layerData.layerStructure.mergeDirection)
		{
		case MergeInput::LayerStructure::mergeDirection_ch:
			{
				for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
				{
					U32 offset = 0;
					for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
					{
						hipError_t err = hipMemcpyAsync(
							&o_lppOutputBuffer[batchNum*this->outputBufferCount + offset],
							&i_lppInputBuffer[inputNum][batchNum*this->lpInputBufferCount[inputNum]],
							sizeof(F32) * this->lpInputBufferCount[inputNum],
							hipMemcpyDeviceToDevice);
						if(err != 0)
							return ErrorCode::ERROR_CODE_CUDA_CALCULATE;

						offset += this->lpInputBufferCount[inputNum];
					}
				}
				hipDeviceSynchronize();
			}
			break;
		case MergeInput::LayerStructure::mergeDirection_x:
			break;
		case MergeInput::LayerStructure::mergeDirection_y:
			break;
		case MergeInput::LayerStructure::mergeDirection_z:
			break;
		}

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** ���͌덷�v�Z�������s����.�w�K�����ɓ��͌덷���擾�������ꍇ�Ɏg�p����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	o_lppDInputBuffer	���͌덷�����i�[�惌�C���[.	[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̗v�f�����K�v.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeInput_GPU::CalculateDInput_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		if(o_lppDInputBuffer)
		{
			// ���͌덷�o�b�t�@�̃A�h���X��z��Ɋi�[
			for(U32 inputNum=0; inputNum<this->GetInputDataCount(); inputNum++)
			{
				this->m_lppDInputBuffer[inputNum] = o_lppDInputBuffer[inputNum];
			}

			switch(this->layerData.layerStructure.mergeDirection)
			{
			case MergeInput::LayerStructure::mergeDirection_ch:
				{
					for(U32 batchNum=0; batchNum<this->GetBatchSize(); batchNum++)
					{
						U32 offset = 0;
						for(U32 inputNum=0; inputNum<this->lpInputBufferCount.size(); inputNum++)
						{
							hipError_t err = hipMemcpyAsync(
								&this->m_lppDInputBuffer[inputNum][batchNum*this->lpInputBufferCount[inputNum]],
								&i_lppDOutputBuffer[batchNum*this->outputBufferCount + offset],
								sizeof(F32) * this->lpInputBufferCount[inputNum],
								hipMemcpyDeviceToDevice);
							if(err != 0)
								return ErrorCode::ERROR_CODE_CUDA_CALCULATE;

							offset += this->lpInputBufferCount[inputNum];
						}
					}
					hipDeviceSynchronize();
				}
				break;
			case MergeInput::LayerStructure::mergeDirection_x:
				break;
			case MergeInput::LayerStructure::mergeDirection_y:
				break;
			case MergeInput::LayerStructure::mergeDirection_z:
				break;
			}
		}
		

		return ErrorCode::ERROR_CODE_NONE;
	}

	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MergeInput_GPU::Training_device(CONST_BATCH_BUFFER_POINTER i_lppInputBuffer[], BATCH_BUFFER_POINTER o_lppDInputBuffer[], CONST_BATCH_BUFFER_POINTER i_lppOutputBuffer, CONST_BATCH_BUFFER_POINTER i_lppDOutputBuffer)
	{
		return this->CalculateDInput_device(i_lppInputBuffer, o_lppDInputBuffer, i_lppOutputBuffer, i_lppDOutputBuffer);
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
