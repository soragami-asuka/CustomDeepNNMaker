#include "hip/hip_runtime.h"
//======================================
// �t�B�[�h�t�H���[�h�j���[�����l�b�g���[�N�̓����������C���[
// �����A������
// GPU�����p
//======================================
#include"stdafx.h"

#include"MaxAveragePooling_DATA.hpp"
#include"MaxAveragePooling_FUNC.hpp"
#include"MaxAveragePooling_Base.h"

#include"MaxAveragePooling_GPU.cuh"
#include"MaxAveragePooling_LayerData_GPU.cuh"

#ifndef __HIPCC__  
    #define __HIPCC__
#endif

#include<device_functions.hpp>

using namespace Gravisbell;
using namespace Gravisbell::Layer::NeuralNetwork;

#define BLOCK_SIZE	(32)

namespace Gravisbell {
namespace Layer {
namespace NeuralNetwork {

	namespace
	{
		// ����p
		__global__ void cuda_func_average_input(const F32* i_lpInputBuffer, F32* o_lpOutputBuffer, const U32 i_inputChSize, U32 i_outputChSize)
		{
			const U32 batchNo = blockIdx.z;
			const U32 chNo    = blockIdx.y;
			const U32 chCount = blockDim.y;

			const U32 bufferPos = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			const U32 outputPos = batchNo * (i_outputChSize * chCount) + chNo * i_outputChSize + bufferPos;
			const U32 inputPos  = batchNo * (i_inputChSize  * chCount) + chNo * i_inputChSize  + bufferPos;

			__shared__ F32 lpTmpBuf[BLOCK_SIZE*2];
			if(inputPos >= i_inputChSize)
				lpTmpBuf[blockIdx.x]  = 0.0f;
			else
				lpTmpBuf[blockIdx.x + 0]  = i_lpInputBuffer[inputPos];
			__syncthreads();

			if(threadIdx.x < 16)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 16];
			__syncthreads();
			if(threadIdx.x < 8)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 8];
			__syncthreads();
			if(threadIdx.x < 4)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 4];
			__syncthreads();
			if(threadIdx.x < 2)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 2];
			__syncthreads();
			if(threadIdx.x < 1)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 1];
			__syncthreads();

			o_lpOutputBuffer[outputPos] = lpTmpBuf[0];
		}
		// �r���v�Z�p(�y��)
		__global__ void cuda_func_average(const F32* i_lpInputBuffer, F32* o_lpOutputBuffer, const U32 i_inputChSize, U32 i_outputChSize)
		{
			const U32 batchNo = blockIdx.z;
			const U32 chNo    = blockIdx.y;
			const U32 chCount = blockDim.y;

			const U32 bufferPos = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			const U32 outputPos = batchNo * (i_outputChSize * chCount) + chNo * i_outputChSize + bufferPos;
			const U32 inputPos  = batchNo * (i_inputChSize  * chCount) + chNo * i_inputChSize  + bufferPos;

			__shared__ F32 lpTmpBuf[BLOCK_SIZE*2];
			lpTmpBuf[blockIdx.x + 0]  = i_lpInputBuffer[inputPos];
			__syncthreads();

			if(threadIdx.x < 16)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 16];
			__syncthreads();
			if(threadIdx.x < 8)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 8];
			__syncthreads();
			if(threadIdx.x < 4)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 4];
			__syncthreads();
			if(threadIdx.x < 2)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 2];
			__syncthreads();
			if(threadIdx.x < 1)
				lpTmpBuf[threadIdx.x] += lpTmpBuf[threadIdx.x + 1];
			__syncthreads();

			o_lpOutputBuffer[outputPos] = lpTmpBuf[0];
		}
	}


	/** �R���X�g���N�^ */
	MaxAveragePooling_GPU::MaxAveragePooling_GPU(Gravisbell::GUID guid, MaxAveragePooling_LayerData_GPU& i_layerData)
		:	MaxAveragePooling_Base	(guid)
		,	layerData						(i_layerData)	/**< ���C���[�f�[�^ */
		,	inputBufferCount				(0)				/**< ���̓o�b�t�@�� */
		,	outputBufferCount				(0)				/**< �o�̓o�b�t�@�� */
		,	m_lppInputBuffer				(NULL)			/**< ���Z���̓��̓f�[�^ */
		,	m_lppDOutputBufferPrev			(NULL)			/**< ���͌덷�v�Z���̏o�͌덷�f�[�^ */
	{
	}
	/** �f�X�g���N�^ */
	MaxAveragePooling_GPU::~MaxAveragePooling_GPU()
	{
	}


	//================================
	// ��{����
	//================================
	/** ���C���[��ʂ̎擾 */
	U32 MaxAveragePooling_GPU::GetLayerKind()const
	{
		return Layer::ELayerKind::LAYER_KIND_GPU | GetLayerKindBase();
	}

	/** ������. �e�j���[�����̒l�������_���ɏ�����
		@return	���������ꍇ0 */
	ErrorCode MaxAveragePooling_GPU::Initialize(void)
	{
		return this->layerData.Initialize();
	}


	//===========================
	// ���C���[�f�[�^�֘A
	//===========================
	/** ���C���[�f�[�^���擾���� */
	MaxAveragePooling_LayerData_Base& MaxAveragePooling_GPU::GetLayerData()
	{
		return this->layerData;
	}
	const MaxAveragePooling_LayerData_Base& MaxAveragePooling_GPU::GetLayerData()const
	{
		return this->layerData;
	}


	//================================
	// ���Z����
	//================================
	/** ���Z�O���������s����.(�w�K�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��PreProcessLearnLoop�ȍ~�̏����͎��s�s��. */
	ErrorCode MaxAveragePooling_GPU::PreProcessLearn(unsigned int batchSize)
	{
		ErrorCode errorCode = this->PreProcessCalculate(batchSize);
		if(errorCode != ErrorCode::ERROR_CODE_NONE)
			return errorCode;

		// ���͍����o�b�t�@���쐬
		this->lpDInputBuffer.resize(this->batchSize * this->inputBufferCount);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z�O���������s����.(���Z�p)
		@param batchSize	�����ɉ��Z���s���o�b�`�̃T�C�Y.
		NN�쐬��A���Z���������s����O�Ɉ�x�����K�����s���邱�ƁB�f�[�^���ƂɎ��s����K�v�͂Ȃ�.
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MaxAveragePooling_GPU::PreProcessCalculate(unsigned int batchSize)
	{
		this->batchSize = batchSize;

		// ���̓o�b�t�@�����m�F
		this->inputBufferCount = this->GetInputBufferCount();
		if(this->inputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_INPUT_COUNT;

		// �o�̓o�b�t�@�����m�F
		this->outputBufferCount = this->GetOutputBufferCount();
		if(this->outputBufferCount == 0)
			return ErrorCode::ERROR_CODE_FRAUD_OUTPUT_COUNT;

		// �o�̓o�b�t�@���쐬
		this->lpOutputBuffer.resize(this->batchSize * this->outputBufferCount);

		// 1CH������̃T�C�Y���v�Z
		this->chSize = this->GetInputDataStruct().x * this->GetInputDataStruct().y * this->GetInputDataStruct().z;

		// �ꎞ�o�b�t�@�̊m��
		this->lpTmpBuffer0.resize((this->chSize + 31)/32*32 * this->GetInputDataStruct().ch * this->batchSize, 0.0f);
		this->lpTmpBuffer1.resize((this->chSize + 31)/32*32 * this->GetInputDataStruct().ch * this->batchSize, 0.0f);

		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K���[�v�̏���������.�f�[�^�Z�b�g�̊w�K�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MaxAveragePooling_GPU::PreProcessLearnLoop(const SettingData::Standard::IData& data)
	{
		if(this->pLearnData != NULL)
			delete this->pLearnData;
		this->pLearnData = data.Clone();

		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}
	/** ���Z���[�v�̏���������.�f�[�^�Z�b�g�̉��Z�J�n�O�Ɏ��s����
		���s�����ꍇ��Calculate�ȍ~�̏����͎��s�s��. */
	ErrorCode MaxAveragePooling_GPU::PreProcessCalculateLoop()
	{
		return Gravisbell::ErrorCode::ERROR_CODE_NONE;
	}


	/** ���Z���������s����.
		@param lpInputBuffer	���̓f�[�^�o�b�t�@. GetInputBufferCount�Ŏ擾�����l�̗v�f�����K�v
		@return ���������ꍇ0���Ԃ� */
	ErrorCode MaxAveragePooling_GPU::Calculate(CONST_BATCH_BUFFER_POINTER i_lpInputBuffer)
	{
		// ���̓o�b�t�@�̃A�h���X���i�[
		this->m_lppInputBuffer = i_lpInputBuffer;

		// ���񏈗�
		U32 tmpInputBufferCount = this->chSize;
		U32 tmpOutputBufferCount = (tmpInputBufferCount + (BLOCK_SIZE-1))/BLOCK_SIZE;
		{
			dim3 grid(tmpOutputBufferCount, this->GetInputDataStruct().ch, this->batchSize);

			cuda_func_average_input<<<grid, BLOCK_SIZE>>>(i_lpInputBuffer, thrust::raw_pointer_cast(&this->lpTmpBuffer0[0]), tmpInputBufferCount, tmpOutputBufferCount);
		}

		while(tmpOutputBufferCount > 1)
		{
			tmpInputBufferCount = tmpOutputBufferCount;
			tmpOutputBufferCount = (tmpInputBufferCount + (BLOCK_SIZE-1))/BLOCK_SIZE;

			cuda_func_average<<<grid, BLOCK_SIZE>>>(i_lpInputBuffer, thrust::raw_pointer_cast(&this->lpTmpBuffer0[0]), tmpInputBufferCount, tmpOutputBufferCount);
		}

		// �eCH�̗v�f��ch�T�C�Y�ŏ��Z���Ė{�̂Ɋi�[




		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �o�̓f�[�^�o�b�t�@���擾����.
		�z��̗v�f����GetOutputBufferCount�̖߂�l.
		@return �o�̓f�[�^�z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER MaxAveragePooling_GPU::GetOutputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpOutputBuffer[0]);
	}
	/** �o�̓f�[�^�o�b�t�@���擾����.
		@param o_lpOutputBuffer	�o�̓f�[�^�i�[��z��. [GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v
		@return ���������ꍇ0 */
	ErrorCode MaxAveragePooling_GPU::GetOutputBuffer(BATCH_BUFFER_POINTER o_lpOutputBuffer)const
	{
		if(o_lpOutputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 outputBufferCount = this->GetOutputBufferCount();

		hipMemcpy(o_lpOutputBuffer, this->GetOutputBuffer(), sizeof(F32)*outputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


	//================================
	// �w�K����
	//================================
	/** �w�K���������s����.
		���͐M���A�o�͐M���͒��O��Calculate�̒l���Q�Ƃ���.
		@param	i_lppDOutputBuffer	�o�͌덷����=�����C���[�̓��͌덷����.	[GetBatchSize()�̖߂�l][GetOutputBufferCount()�̖߂�l]�̗v�f�����K�v.
		���O�̌v�Z���ʂ��g�p���� */
	ErrorCode MaxAveragePooling_GPU::Training(CONST_BATCH_BUFFER_POINTER i_lppDOutputBufferPrev)
	{
		// �o�͌덷�o�b�t�@�̃A�h���X��z��Ɋi�[
		this->m_lppDOutputBufferPrev = i_lppDOutputBufferPrev;


		return ErrorCode::ERROR_CODE_NONE;
	}


	/** �w�K�������擾����.
		�z��̗v�f����[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]
		@return	�덷�����z��̐擪�|�C���^ */
	CONST_BATCH_BUFFER_POINTER MaxAveragePooling_GPU::GetDInputBuffer()const
	{
		return thrust::raw_pointer_cast(&this->lpDInputBuffer[0]);
	}
	/** �w�K�������擾����.
		@param lpDInputBuffer	�w�K�������i�[����z��.[GetBatchSize()�̖߂�l][GetInputBufferCount()�̖߂�l]�̔z�񂪕K�v */
	ErrorCode MaxAveragePooling_GPU::GetDInputBuffer(BATCH_BUFFER_POINTER o_lpDInputBuffer)const
	{
		if(o_lpDInputBuffer == NULL)
			return ErrorCode::ERROR_CODE_COMMON_NULL_REFERENCE;

		const U32 batchSize = this->GetBatchSize();
		const U32 inputBufferCount = this->GetInputBufferCount();

		hipMemcpy(o_lpDInputBuffer, this->GetDInputBuffer(), sizeof(F32)*inputBufferCount*batchSize, hipMemcpyDeviceToHost);

		return ErrorCode::ERROR_CODE_NONE;
	}


} // Gravisbell;
} // Layer;
} // NeuralNetwork;
